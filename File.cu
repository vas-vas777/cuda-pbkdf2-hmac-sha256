#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//#include "HMAC-SHA256.cpp"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <openssl/hmac.h>
#include <openssl/evp.h>
#include <string>
#include <fstream>

#define N 2048
#define M 2048


#define uchar unsigned char
#define uint unsigned int

#define DBL_INT_ADD(a,b,c) if (a > 0xffffffff - (c)) ++b; a += c;
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

typedef struct {
    uchar data[64];
    uint datalen;
    uint bitlen[2];
    uint state[8];
} SHA256_CTX;

uint k[64] = {
    0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
    0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
    0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
    0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
    0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
    0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
    0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
    0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

__device__ void SHA256Transform(SHA256_CTX* ctx, uchar data[])
{
    uint a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

    for (i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
    for (; i < 64; ++i)
        m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

    a = ctx->state[0];
    b = ctx->state[1];
    c = ctx->state[2];
    d = ctx->state[3];
    e = ctx->state[4];
    f = ctx->state[5];
    g = ctx->state[6];
    h = ctx->state[7];

    for (i = 0; i < 64; ++i) {
        t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
        t2 = EP0(a) + MAJ(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    ctx->state[0] += a;
    ctx->state[1] += b;
    ctx->state[2] += c;
    ctx->state[3] += d;
    ctx->state[4] += e;
    ctx->state[5] += f;
    ctx->state[6] += g;
    ctx->state[7] += h;
}

__device__ void SHA256Init(SHA256_CTX* ctx)
{
    ctx->datalen = 0;
    ctx->bitlen[0] = 0;
    ctx->bitlen[1] = 0;
    ctx->state[0] = 0x6a09e667;
    ctx->state[1] = 0xbb67ae85;
    ctx->state[2] = 0x3c6ef372;
    ctx->state[3] = 0xa54ff53a;
    ctx->state[4] = 0x510e527f;
    ctx->state[5] = 0x9b05688c;
    ctx->state[6] = 0x1f83d9ab;
    ctx->state[7] = 0x5be0cd19;
}

__device__ void SHA256Update(SHA256_CTX* ctx, uchar data[], uint len)
{
    for (uint i = 0; i < len; ++i) {
        ctx->data[ctx->datalen] = data[i];
        ctx->datalen++;
        if (ctx->datalen == 64) {
            SHA256Transform(ctx, ctx->data);
            DBL_INT_ADD(ctx->bitlen[0], ctx->bitlen[1], 512);
            ctx->datalen = 0;
        }
    }
}

__device__ void SHA256Final(SHA256_CTX* ctx, uchar hash[])
{
    uint i = ctx->datalen;

    if (ctx->datalen < 56) {
        ctx->data[i++] = 0x80;
        while (i < 56)
            ctx->data[i++] = 0x00;
    }
    else {
        ctx->data[i++] = 0x80;
        while (i < 64)
            ctx->data[i++] = 0x00;
        SHA256Transform(ctx, ctx->data);
        memset(ctx->data, 0, 56);
    }

    DBL_INT_ADD(ctx->bitlen[0], ctx->bitlen[1], ctx->datalen * 8);
    ctx->data[63] = ctx->bitlen[0];
    ctx->data[62] = ctx->bitlen[0] >> 8;
    ctx->data[61] = ctx->bitlen[0] >> 16;
    ctx->data[60] = ctx->bitlen[0] >> 24;
    ctx->data[59] = ctx->bitlen[1];
    ctx->data[58] = ctx->bitlen[1] >> 8;
    ctx->data[57] = ctx->bitlen[1] >> 16;
    ctx->data[56] = ctx->bitlen[1] >> 24;
    SHA256Transform(ctx, ctx->data);

    for (i = 0; i < 4; ++i) {
        hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
    }
}

__device__ char* SHA256(unsigned char* data) {
    int strLen = strlen((char*)data);
    SHA256_CTX ctx;
    unsigned char hash[32];
    char* hashStr = malloc(65);
    strcpy(hashStr, "");

    SHA256Init(&ctx);
    SHA256Update(&ctx, data, strLen);
    SHA256Final(&ctx, hash);

    char s[3];
    for (int i = 0; i < 32; i++) {
        sprintf(s, "%02x", hash[i]);
        strcat(hashStr, s);
    }

    return hashStr;
}


hipError_t addWithCuda(std::string passwords, unsigned char* salt, int length_salt, int iterations, int length_hash, unsigned char* hash_output, unsigned char* pass_output);
//hipError_t readWithCuda(std::ifstream file, std::string )


__global__ void addKernel(char* passwords, unsigned char* salt, int length_salt, int iterations, int length_hash, unsigned char* hash_output, unsigned char* pass_output)
{
    
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    //PKCS5_PBKDF2_HMAC(&passwords[i], strlen(&passwords[i]), salt, strlen((char*)salt), iterations, EVP_sha256(), length_hash, &hash_output[length_hash]);
    
    //pbkdf2_hmac_sha256(i, passwords, salt, length_salt, iterations, length_hash, hash_output, pass_output);
    //PKCS5_PBKDF2_HMAC(password, strlen(password), salt, strlen((char*)salt), 4096, EVP_sha256(), i, hash_output);
    //PKCS5_PBKDF2_HMAC(password, strlen(password), salt, strlen((char*)salt), 4096, EVP_sha256(), i, hash_output);
    //PKCS5_PBKDF2_HMAC(password, strlen(password), salt, strlen((char*)salt), 4096, EVP_sha256(), i, hash_output);
    //PKCS5_PBKDF2_HMAC(password, strlen(password), salt, strlen((char*)salt), 4096, EVP_sha256(), i, hash_output);
    //PKCS5_PBKDF2_HMAC(password, strlen(password), salt, strlen((char*)salt), 4096, EVP_sha256(), i, hash_output);
}

//__global__ void read_from_file(std::ifstream file)

int main()
{
    /*const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };*/
    //const int N = 2048; //������
    //const int M = 2048; //�������
    //����� ����
    unsigned char hash_unknowed_password[] = "127198299418516314213416115420814586671341501494113814733145806219619820859207235214206";
    const int hash_length = 32;
    unsigned char* output_pass;
    unsigned char salt[] = "POINT";
    //unsigned char *password;
    //int iterations = 4096;
    unsigned char hash_output[hash_length];
    std::ifstream file_of_passwords;
    file_of_passwords.open("E:\\repos\\9mil.txt");
    std::string passwords;
    std::string pass;
    if (passwords.empty()) 
    {
        while (!file_of_passwords.eof())
        {
            file_of_passwords >> pass;
            passwords.append(pass);
        }
    }
    //std::cout << passwords << std::endl;
    // Add vectors in parallel.
    hipError_t cudaStatus;

    while (hash_output != hash_unknowed_password)
    {
        
        cudaStatus = addWithCuda(passwords, salt, strlen((char*)salt), 4096, hash_length, hash_output, output_pass);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }
    }
    std::cout << output_pass << std::endl;
    /*printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(std::string passwords, unsigned char* salt, int length_salt, int iterations, int length_hash, unsigned char* hash_output, unsigned char* pass_output)
{

    //const int len_of_hash = 32;
    unsigned char *temp_hash_output;
    //const char password[] = "12345";
    unsigned char *temp_salt;
    //char* passwords;
    char* temp_passwords;
    char* temp_passwords_for_string;
    unsigned char* temp_pass_output;
    std::copy(passwords.begin(), passwords.end(), temp_passwords_for_string);
    //std::ifstream file_of_passwords;

    //std::cout << strlen((char*)salt) << std::endl;

   

    //std::cout << "\n" << std::endl;
    //std::cout << "Hello World!\n";
    //return *hash;

    /*int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;*/
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&temp_salt, length_salt * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&temp_passwords, strlen(temp_passwords) * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&temp_hash_output, length_hash * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&temp_passwords, strlen(temp_passwords) * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(temp_salt, salt, length_salt * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(temp_passwords, temp_passwords_for_string, strlen(temp_passwords_for_string) * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(temp_hash_output, hash_output, strlen((char*)hash_output) * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(temp_pass_output, pass_output, strlen((char*)pass_output) * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    /*cudaStatus = hipMemcpy(temp_hash_output, , size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/

    // Launch a kernel on the GPU with one thread for each element.

    addKernel <<<N, M >>> (temp_passwords, temp_salt, length_salt, iterations, length_hash, temp_hash_output, temp_pass_output);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(hash_output, temp_hash_output, length_hash * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(pass_output, temp_pass_output, length_hash * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(temp_hash_output);
    hipFree(temp_salt);
    hipFree(temp_passwords);
    return cudaStatus;
}

//__host__ __location__(device) int PKCS5_PBKDF2_HMAC(const char* pass, int passlen, const unsigned char* salt, int saltlen, int iter, const EVP_MD* digest, int keylen, unsigned char* out)
//{
//    return PKCS5_PBKDF2_HMAC(pass, passlen, salt, saltlen, iter, digest, keylen, out);
//}
//
//__host__ __location__(device)size_t strlen(const char* arr)
//{
//    return strlen(arr);
//}
//
//__host__ __location__(device)EVP_MD* EVP_sha256(void)
//{
//    return nullptr;
//}


//__host__ __device__ int PKCS5_PBKDF2_HMAC(const char* pass, int passlen, const unsigned char* salt, int saltlen, int iter, const EVP_MD* digest, int keylen, unsigned char* out)
//{
//    return int();
//}
//
//__host__ __device__ size_t strlen(const char*)
//{
//    return size_t();
//}
//
//__host__ __device__ const EVP_MD* EVP_sha256(void)
//{
//    return nullptr;
//}




