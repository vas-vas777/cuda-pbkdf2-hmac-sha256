﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string>
#include <bitset>
#include <iostream>


const uint32_t round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };


hipError_t addWithCuda(char* binary_pass, int length_block);

__host__ uint32_t* to_binary_32bit(uint32_t number)
{
    uint32_t* binary_str = new uint32_t[32]{ 0 };


    int count = 31;
    //std::cout << number << std::endl;
    while (number!=0)
    {
        /*std::cout << "binary32" << std::endl;
        std::cout << count << std::endl;*/
        binary_str[count] = number % 2;
        
        number /= 2;
        count--;
       
    }
   
    
    return binary_str;
    //delete[] binary_str;
}

__host__ uint32_t str_to_32bitnumber(uint32_t* str)
{
    uint32_t number1 = 0;
    for (auto i = 0; i < 32; ++i)
    {
        number1 += str[i] * pow(2, (31 - i));
    }
    return number1;
}

__host__ uint32_t* and_strs_32bit(uint32_t* str1, uint32_t* str2)
{
    uint32_t* result_str = new uint32_t[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = str1[i] & str2[i];
    }
    return result_str;
    //delete[] result_str;
}

__host__ uint32_t* inverse_str_32bit(uint32_t* str1)
{
    uint32_t* result_str = new uint32_t[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = !str1[i];
        //std::cout << !str1[i];
    }
   // std::cout << std::endl;
    return result_str;
    //delete[] result_str;
}

__host__ uint32_t sum_strs_32bit(uint32_t* str1, uint32_t* str2)
{
    //uint32_t* result_sum = new uint32_t[32]{ 0 };
    uint32_t number1 = 0;
    uint32_t number2 = 0;
    uint32_t res_number = 0;
    for (auto i = 0; i < 32; ++i)
    {
        number1 += str1[i] * pow(2, (31 - i));
    }
    for (auto i = 0; i < 32; ++i)
    {
        number2 += str2[i] * pow(2, (31 - i));
    }
    //std::cout << number1 << std::endl;
    //std::cout << number2 << std::endl;
    res_number = (number1 + number2) % 4294967296;

    //std::cout << number2 << std::endl;
    return res_number;
   // delete[]result_sum;
    //delete[]result_sum;
}

__host__ uint32_t* xor_strs_32bit(uint32_t* str1, uint32_t* str2)
{
    uint32_t* result_xor = new uint32_t[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_xor[i] = str1[i] ^ str2[i];
    }
    return result_xor;
    //delete[]result_xor;
}

__host__ uint32_t rigth_rotate(uint32_t* str, unsigned int num)
{
    uint32_t number = 0;
    uint32_t* rotated_str = new uint32_t[32]{ 0 };
    memcpy(rotated_str, str, 4 * 32);
    number = str_to_32bitnumber(rotated_str);
    return (number >> num | number << (32 - num));
    //_rotr64(12345668877, 7);

   /* std::cout << "begin_rotation" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << rotated_str[i];
    }
    std::cout << std::endl;*/
    //for (unsigned int count = 1; count <= num; ++count)
    //{
    //    last = rotated_str[31];
    //    //std::cout << count << std::endl;
    //    for (auto i = 31; i > 0; --i)
    //    {
    //        rotated_str[i] = rotated_str[i - 1];
    //    }
    //    rotated_str[0] = last;
    //    
    //}
    //std::cout << "rotated_str=" << num << std::endl;
    //for (auto i = 0; i < 32; ++i)
    //{
    //    std::cout << rotated_str[i];
    //}
    //std::cout << std::endl;
    //return rotated_str;
    //delete[] rotated_str;
}

__host__ uint32_t rigth_shift(uint32_t* str, unsigned int num)
{
    //uint32_t last = 0;
   // for (auto count = 0; count < num; ++count)
   // {
       // last = str[31];
    uint32_t* shifted_str = new uint32_t[32]{ 0 };
    memcpy(shifted_str, str, 4 * 32);
    uint32_t number = str_to_32bitnumber(shifted_str);
    return number >> num;
       /* for (unsigned int i = 31; i > 0; --i)
        {
            if (i >= num)
            {
                shifted_str[i] = shifted_str[i - num];
            }
            else
            {
                shifted_str[i] = 0;
            }
        }*/
       // str[0] = last;
  //  }
    /*    std::cout << "right_shift="<<num << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << shifted_str[i];
    }
    std::cout  << std::endl;*/
    //return shifted_str;
    //delete[] shifted_str;
}



__host__ uint32_t* password_xor_with_IPAD(char* password,int length)
{
    uint32_t* binary_str = new uint32_t[512]{ 0 };
    uint32_t IPAD[] = { 0,0,1,1,0,1,1,0 };
    uint32_t* ipad = new uint32_t[512]{ 0 };
    for (auto i = 0; i < 512; ++i)
    {
        if (i < 8 * length)
        {
            binary_str[i] = (0 != (password[i / 8] & 1 << (~i & 7)));
        }
        else
        {
            binary_str[i] = 0;
        }
    }
    std::cout << "password" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        ipad[i] = IPAD[i % 8];
        
    }
    for (auto i = 0; i < 512; ++i)
    {
        binary_str[i] = binary_str[i] ^ ipad[i];
    }
    std::cout << "password_xor_with_IPAD 512 bits" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    //std::cout << sizeof(binary_str) << std::endl;
    
    return binary_str;
  
    
}

__host__ uint32_t* password_xor_with_OPAD(char* password, int length)
{
    uint32_t* binary_str = new uint32_t[512]{ 0 };
    uint32_t OPAD [] = { 0,1,0,1,1,1,0,0 };
    uint32_t* opad = new uint32_t[512]{ 0 };
    for (auto i = 0; i < 512; ++i)
    {
        if (i < 8 * length)
        {
            binary_str[i] = (0 != (password[i / 8] & 1 << (~i & 7)));
        }
        else
        {
            binary_str[i] = 0;
        }
    }
    for (auto i = 0; i < 512; ++i)
    {
        opad[i] = OPAD[i % 8];

    }
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << opad[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        binary_str[i] = binary_str[i] ^ opad[i];
    }
   /* for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }*/

    
    return binary_str;
    delete[] binary_str;
    delete[] opad;
    
}

 __host__ uint32_t* preparation_sha256_with_IPAD(uint32_t* password_xor_with_ipad, uint32_t*prev_hash)
{
     uint32_t* binary_str = new uint32_t[1024]{ 0 };
     memcpy(binary_str, password_xor_with_ipad, 4 * 512);
     memcpy(binary_str + 512, prev_hash, 4 * 256);
    binary_str[768] = 1;
    binary_str[1014] = 1;
    binary_str[1015] = 1;
    //memmove(binary_str, password_xor_with_ipad, 512);
    /*std::cout << "preparation_sha256_with_IPAD-1024 bits" << std::endl;
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;*/
    
   
    //for (auto i = 0; i < 2048; ++i)
    //{
    //    std::cout << output_message[i];
    //}
    //std::cout << std::endl;
    
    return binary_str;
    
}

__host__ uint32_t* preparation_sha256_with_OPAD(uint32_t* password_xor_with_opad, uint32_t* prev_hash)
{
    uint32_t* binary_str = new uint32_t[1024]{ 0 };
   // uint32_t* output_message = new uint32_t[2048]{ 0 };
    //memcpy(binary_str, password_xor_with_ipad, 512);
    /*std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << password_xor_with_opad[i];
    }
    std::cout << std::endl;*/

    memcpy(binary_str, password_xor_with_opad, 4 * 512);
    memcpy(binary_str + 512, prev_hash, 4 * 256);
    binary_str[768] = 1;
    binary_str[1014] = 1;
    binary_str[1015] = 1;
   
    return binary_str;
   
}

__host__ uint32_t* main_loop_sha256_with_ipad(uint32_t* message)
{
    uint32_t* h0 = new uint32_t[] { 0, 1, 1, 0,  1, 0, 1, 0,  0, 0, 0, 0,  1, 0, 0, 1,  1, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 1 };
    uint32_t* h1 = new uint32_t[] { 1, 0, 1, 1,  1, 0, 1, 1,  0, 1, 1, 0,  0, 1, 1, 1,  1, 0, 1, 0,  1, 1, 1, 0,  1, 0, 0, 0,  0, 1, 0, 1 };
    uint32_t* h2 = new uint32_t[] { 0, 0, 1, 1,  1, 1, 0, 0,  0, 1, 1, 0,  1, 1, 1, 0,  1, 1, 1, 1,  0, 0, 1, 1,  0, 1, 1, 1,  0, 0, 1, 0 };
    uint32_t* h3 = new uint32_t[] { 1, 0, 1, 0,  0, 1, 0, 1,  0, 1, 0, 0,  1, 1, 1, 1,  1, 1, 1, 1,  0, 1, 0, 1,  0, 0, 1, 1,  1, 0, 1, 0 };
    uint32_t* h4 = new uint32_t[] { 0, 1, 0, 1,  0, 0, 0, 1,  0, 0, 0, 0,  1, 1, 1, 0,  0, 1, 0, 1,  0, 0, 1, 0,  0, 1, 1, 1,  1, 1, 1, 1 };
    uint32_t* h5 = new uint32_t[] { 1, 0, 0, 1,  1, 0, 1, 1,  0, 0, 0, 0,  0, 1, 0, 1,  0, 1, 1, 0,  1, 0, 0, 0,  1, 0, 0, 0,  1, 1, 0, 0 };
    uint32_t* h6 = new uint32_t[] { 0, 0, 0, 1,  1, 1, 1, 1,  1, 0, 0, 0,  0, 0, 1, 1,  1, 1, 0, 1,  1, 0, 0, 1,  1, 0, 1, 0,  1, 0, 1, 1 };
    uint32_t* h7 = new uint32_t[] { 0, 1, 0, 1,  1, 0, 1, 1,  1, 1, 1, 0,  0, 0, 0, 0,  1, 1, 0, 0,  1, 1, 0, 1,  0, 0, 0, 1,  1, 0, 0, 1 };
    uint32_t* part_message1 = new uint32_t[512]{ 0,1,1,0,1,0,0,0, 0,1,1,0,0,1,0,1, 0,1,1,0,1,1,0,0, 0,1,1,0,1,1,0,0, 0,1,1,0,1,1,1,1, 0,0,1,0,0,0,0,0, 0,1,1,1,0,1,1,1, 0,1,1,0,1,1,1,1,
0,1,1,1,0,0,1,0, 0,1,1,0,1,1,0,0, 0,1,1,0,0,1,0,0, 1,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,1,0,1,1,0,0,0 };
    uint32_t* part_message2 = new uint32_t[512]{ 0 };
   // memcpy(part_message1, message, 4 * 512);
    memcpy(part_message2, message + 512, 4 * 512);
    /*std::cout << "message" << std::endl;
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << message[i];
    }
    std::cout << std::endl;*/
    std::cout << "part_message1" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << part_message1[i];
    }
    std::cout << std::endl;
   /* std::cout << std::endl;
    std::cout << "part_message2" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << part_message2[i];
    }
    std::cout << std::endl;*/
    int count = 1;//счётчик для 2ух итераций
   

   
    while (count < 2)
    {
        
        
        uint32_t* S1 = new uint32_t[32]{ 0 };
        uint32_t* ch = new uint32_t[32]{ 0 };
        uint32_t* temp1 = new uint32_t[32]{ 0 };
        uint32_t* S0 = new uint32_t[32]{ 0 };
        uint32_t* maj = new uint32_t[32]{ 0 };
        uint32_t* temp2 = new uint32_t[32]{ 0 };

        
        uint32_t sum_ext2_s1 = 0;

        uint32_t* a = new uint32_t[32]{ 0 };
        uint32_t* b = new uint32_t[32]{ 0 };
        uint32_t* c = new uint32_t[32]{ 0 };
        uint32_t* d = new uint32_t[32]{ 0 };
        uint32_t* e = new uint32_t[32]{ 0 };
        uint32_t* f = new uint32_t[32]{ 0 };
        uint32_t* g = new uint32_t[32]{ 0 };
        uint32_t* h = new uint32_t[32]{ 0 };
        uint32_t** extend_part_message1 = new uint32_t* [64]{ 0 };
        for (auto i = 0; i < 64; ++i)
        {
            extend_part_message1[i] = new uint32_t[32]{ 0 };
        }

        if (count == 1)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message1 + (i * 32), 4*32);
            }
        }

        if (count == 2)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message2 + (i * 32), 4*32);
            }
        }
       /* std::cout << "extend_part_message" << std::endl;
        for (auto i = 0; i < 64; ++i)
        {
            std::cout << i << "-";
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;*/

        for (auto i = 16; i < 64; ++i)
        {
            uint32_t* s0 = new uint32_t[32]{ 0 };
            uint32_t* s1 = new uint32_t[32]{ 0 };
            uint32_t sum_ext1_2_s0_s1 = 0;
            std::cout << "start-i=" <<i<<"i-15="<<i-15<< std::endl;
            memcpy(s0, xor_strs_32bit(xor_strs_32bit(to_binary_32bit(rigth_rotate(extend_part_message1[i - 15], 7)),
                to_binary_32bit(rigth_rotate(extend_part_message1[i - 15], 18))),
                to_binary_32bit(rigth_shift(extend_part_message1[i - 15], 3))), 4*32);
            std::cout << "s0 - " << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << s0[j];
            }
            std::cout << std::endl;
            std::cout << "start2-i=" << i << "i-2=" << i - 2 << std::endl;
            memcpy(s1, xor_strs_32bit(xor_strs_32bit(to_binary_32bit(rigth_rotate(extend_part_message1[i - 2], 17)),
                to_binary_32bit(rigth_rotate(extend_part_message1[i - 2], 19))),
                to_binary_32bit(rigth_shift(extend_part_message1[i - 2], 10))), 4*32);
            std::cout << "s1 - " << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << s1[j];
            }
            std::cout << std::endl;
            std::cout << "exte[i-16] - i-16="<<i-16 << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i-16][j];
            }
            std::cout << std::endl;
            std::cout << "exte[i-7] - i-7=" << i - 7 << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i - 7][j];
            }
            std::cout << std::endl;
        
            sum_ext1_2_s0_s1 = (sum_strs_32bit(extend_part_message1[i - 16], s0) + sum_strs_32bit(extend_part_message1[i - 7], s1)) % 4294967296;
            //std::cout << "i-" << i << " " << sum_ext1_2_s0_s1 << std::endl;
            //std::cout << std::endl;
           
            memcpy(extend_part_message1[i], to_binary_32bit(sum_ext1_2_s0_s1), 4*32);
           /* std::cout << "exte[" << i << "]" << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;*/
           
        }
        std::cout << "extend_part_message_after" << std::endl;
        for (auto i = 0; i < 64; ++i)
        {
            std::cout << i << "- ";
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        /*std::cout << "a" << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << a[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << b[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << c[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << d[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << e[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << f[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << g[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << h[i];
        }
        std::cout << std::endl;*/
        memcpy(a, h0, 4*32);
        memcpy(b, h1, 4*32);
        memcpy(c, h2, 4*32);
        memcpy(d, h3, 4*32);
        memcpy(e, h4, 4*32);
        memcpy(f, h5, 4*32);
        memcpy(g, h6, 4*32);
        memcpy(h, h7, 4*32);
       /* std::cout << "a" << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << a[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << b[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << c[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << d[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << e[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << f[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << g[i];
        }
        std::cout << std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << h[i];
        }
        std::cout << std::endl;*/
        for (auto i = 0; i < 64; ++i)
        {
         //   S1 = xor_strs_32bit(xor_strs_32bit(rigth_rotate(e, 6), rigth_rotate(e, 11)),
           //     rigth_rotate(e, 25));
            memcpy(S1, xor_strs_32bit(xor_strs_32bit(to_binary_32bit(rigth_rotate(e, 6)), to_binary_32bit(rigth_rotate(e, 11))),
                to_binary_32bit(rigth_rotate(e, 25))), 4*32);
            memcpy(ch, xor_strs_32bit(and_strs_32bit(e, f), and_strs_32bit(inverse_str_32bit(e), g)), 4*32);
           // ch = xor_strs_32bit(and_strs_32bit(e, f), and_strs_32bit(inverse_str_32bit(e), g));
            memcpy(temp1, to_binary_32bit((sum_strs_32bit(h, S1) + sum_strs_32bit(ch, extend_part_message1[i]) + round_consts[i]) % 4294967296), 4*32);
           // temp1 = to_binary_32bit((sum_strs_32bit(h, S1) + sum_strs_32bit(ch, extend_part_message1[i]) + round_consts[i]) % 4294967296);
            memcpy(S0, xor_strs_32bit(xor_strs_32bit(to_binary_32bit(rigth_rotate(a, 2)), to_binary_32bit(rigth_rotate(a, 13))),
                to_binary_32bit(rigth_rotate(a, 22))), 4*32);
           // S0 = xor_strs_32bit(xor_strs_32bit(rigth_rotate(a, 2), rigth_rotate(a, 13)),
             //   rigth_rotate(a, 22));
            memcpy(maj, xor_strs_32bit(xor_strs_32bit(and_strs_32bit(a, b), and_strs_32bit(a, c)),
                and_strs_32bit(b, c)), 4*32);
           // maj = xor_strs_32bit(xor_strs_32bit(and_strs_32bit(a, b), and_strs_32bit(a, c)),
             //   and_strs_32bit(b, c));
            memcpy(temp2, to_binary_32bit(sum_strs_32bit(S0, maj)), 4*32);
           // temp2 = to_binary_32bit(sum_strs_32bit(S0, maj));
            memcpy(h, g, 4*32);
            //h = g;
            memcpy(g, f, 4*32);
           // g = f;
           // f = e;
            memcpy(f, e, 4*32);
            memcpy(e, to_binary_32bit(sum_strs_32bit(d, temp1)), 4*32);
            //e = to_binary_32bit(sum_strs_32bit(d, temp1));
            memcpy(d, c, 4*32);
            memcpy(c, b, 4*32);
            memcpy(b, a, 4*32);
           /* d = c;
            c = b;
            b = a;*/
            memcpy(a, to_binary_32bit(sum_strs_32bit(temp1, temp2)), 4*32);
            //a = to_binary_32bit(sum_strs_32bit(temp1, temp2));
        }
        count++;
       
        memcpy(h0, to_binary_32bit(sum_strs_32bit(h0, a)), 4*32);
        memcpy(h1, to_binary_32bit(sum_strs_32bit(h1, b)), 4*32);
        memcpy(h2, to_binary_32bit(sum_strs_32bit(h2, c)), 4*32);
        memcpy(h3, to_binary_32bit(sum_strs_32bit(h3, d)), 4*32);
        memcpy(h4, to_binary_32bit(sum_strs_32bit(h4, e)), 4*32);
        memcpy(h5, to_binary_32bit(sum_strs_32bit(h5, f)), 4*32);
        memcpy(h6, to_binary_32bit(sum_strs_32bit(h6, g)), 4*32);
        memcpy(h7, to_binary_32bit(sum_strs_32bit(h7, h)), 4*32);
        /*delete[] extend_part_message1;
       delete[] s0;
       delete[] s1;
       delete[] S0;
       delete[] S1;
       delete[] ch;
       delete[] temp1;
       delete[] temp2;
       delete[] maj;*/
        /*h0 = to_binary_32bit(sum_strs_32bit(h0, a));
        h1 = to_binary_32bit(sum_strs_32bit(h1, b));
        h2 = to_binary_32bit(sum_strs_32bit(h2, c));
        h3 = to_binary_32bit(sum_strs_32bit(h3, d));
        h4 = to_binary_32bit(sum_strs_32bit(h4, e));
        h5 = to_binary_32bit(sum_strs_32bit(h5, f));
        h6 = to_binary_32bit(sum_strs_32bit(h6, g));
        h7 = to_binary_32bit(sum_strs_32bit(h7, h));*/
        
    }
    std::cout << "h0" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    std::cout << "h1" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h1[i];
    }
    std::cout << std::endl;
    std::cout << "h2" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h2[i];
    }
    std::cout << std::endl;
    std::cout << "h3" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h3[i];
    }
    std::cout << std::endl;
    std::cout << "h4" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h4[i];
    }
    std::cout << std::endl;
    std::cout << "h5" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h5[i];
    }
    std::cout << std::endl;
    std::cout << "h6" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h6[i];
    }
    std::cout << std::endl;
    std::cout << "h7" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h7[i];
    }
    std::cout << std::endl;
    uint32_t* hash = new uint32_t[256]{ 0 };
    
    memcpy(hash, h0, 4*32);
    memcpy(hash + 32, h1, 4*32);
    memcpy(hash + 64, h2, 4*32);
    memcpy(hash + 96, h3, 4*32);
    memcpy(hash + 128, h4, 4*32);
    memcpy(hash + 160, h5, 4*32);
    memcpy(hash + 192, h6, 4*32);
    memcpy(hash + 224, h7, 4*32);
    return hash;
}

__global__ void addKernel(char* binary_pass, int length_block)
{
    //str_to_32bitnumber(binary_pass);
    


}
int main()
{
   
    std::string password = "Hello world";
    uint32_t* password_xor_with_ipad = new uint32_t[512]{ 0 };
    memcpy(password_xor_with_ipad, password_xor_with_IPAD("1234", 4), 4 * 512);
    uint32_t* prev_hash_u = new uint32_t[256]{ 0 };
    uint32_t* messsage = new uint32_t[1024]{ 0 };
    memcpy(messsage, preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash_u), 4 * 1024);

   // uint32_t* messsage=preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash_u);
    uint32_t* hash = new uint32_t[256]{ 0 };
    memcpy(hash, main_loop_sha256_with_ipad(messsage), 4 * 256);
    //uint32_t* hash=main_loop_sha256_with_ipad(messsage);
    std::cout << "hash" << std::endl;
    for (auto i = 0; i < 256; ++i)
    {
        std::cout << hash[i];
    }
  /* uint32_t* h0 = new uint32_t[] { 1, 0, 0, 0, 0, 1, 1, 0, 1, 1, 0, 1, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 1 };
   uint32_t* res = new uint32_t[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    std::cout << sizeof(uint32_t) << std::endl;
    memcpy(res, to_binary_32bit(rigth_rotate(h0, 10)), 4 * 32);
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << res[i];
    }
    std::cout << std::endl;
    std::cout << rigth_rotate(h0, 10) << std::endl;*/
   /* uint32_t* h0 = new uint32_t[] { 0, 1, 1, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1 };
    uint32_t* h1 = new uint32_t[] { 1, 0, 1, 1, 1, 0, 1, 1, 0, 1, 1, 0, 0, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 1, 0, 1 };
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h1[i];
    }
    std::cout << std::endl;
    uint32_t* prev_hash_u;
    uint32_t number = 0;
    prev_hash_u = rigth_rotate(h0, 32);
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << prev_hash_u[i];
    }*/
   // std::wcout << number << std::endl;
   
    /*
    uint32_t* h0 = new uint32_t[] { 0, 1, 1, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1 };
    uint32_t* h1 = new uint32_t[] { 1, 0, 1, 1, 1, 0, 1, 1, 0, 1, 1, 0, 0, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 1, 0, 1 };
    uint32_t* res = new uint32_t[32];
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    res = rigth_shift(h0, 5);
    
    std::cout << "res" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << res[i];
    }*/
    //password_xor_with_IPAD("1234", 4);
    // Add vectors in parallel.
    /*cudaError_t cudaStatus = addWithCuda(bin_str_512, 512);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }*/

   /* printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/
   

    //// cudaDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = cudaDeviceReset();
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(char* binary_pass, int length_block)
{
    /*int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;*/
    //char* dev_hash_output = "1111";
    char* dev_binary_pass = "111";
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_binary_pass, length_block *sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

   /* cudaStatus = cudaMalloc((void**)&dev_hash_output, length_block * sizeof(char));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }*/

    /*cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }*/

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_binary_pass, binary_pass, length_block * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

   /* cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }*/

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<16, 16 >>>(dev_binary_pass,length_block);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(binary_pass, dev_binary_pass, length_block * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_binary_pass);
   // cudaFree(dev_hash_output);
    //cudaFree(dev_b);
    
    return cudaStatus;
}
