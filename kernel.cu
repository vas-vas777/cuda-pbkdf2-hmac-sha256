#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <random>
#include <string>
#include <bitset>
#include <iostream>



//#include <openssl/hmac.h>




__constant__ const uint32_t round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };



//__device__ uint32_t and_result[32]{ 1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1 };
//__device__ uint32_t inverse_result[32]{ 0 };
//__device__ uint32_t result_xor2[32]{ 0 };
//__device__ uint32_t binary_str[32]{ 0 };







__device__ uint32_t* to_binary_32bit(uint32_t number, uint32_t* binary_str)
{
    
   // uint32_t binary_str[32]{ 0 };
   
    int count = 31;
    while (number != 0)
    {

        binary_str[count] = number % 2;

        number /= 2;
        count--;

    }
    
  return binary_str;
   // delete[] binary_str;
}

__device__ uint32_t str_to_32bitnumber(uint32_t* str)
{
    uint32_t number1 = 0;
    //
   // int i = threadIdx.x ;
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (auto i = 0; i < 32; ++i)
    {
        uint32_t number2 = 1;
        for (auto j = 0; j < (31 - i); ++j)
        {
            number2 *= 2;
        }
        number1 += str[i] * number2;
    }

    return number1;
}

__device__ uint32_t* and_strs_32bit(uint32_t* str1, uint32_t* str2, uint32_t* and_result)
{
   
   // sha256_struct* ctr;
    //uint32_t and_result[32]{ 1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1,  1, 1, 1, 1 };;
   
    for (auto i = 0; i < 32; ++i)
    {
       /* atomicAnd(&and_result[i], str1[i]);
        atomicAnd(&and_result[i], str2[i]);*/
       // __syncthreads();
      and_result[i] = str1[i] & str2[i];
    }
    
   return and_result;
   
}

__device__ uint32_t* inverse_str_32bit(uint32_t* str1, uint32_t* inverse_result)
{
    
  //  uint32_t inverse_result[32]{ 0 };
    //sha256_struct* ctr;
    for (auto i = 0; i < 32; ++i)
    {
        inverse_result[i] = !(str1[i]);
    }
    
   return inverse_result;
   // delete[] inverse_result;
}



__device__ uint32_t sum_strs_32bit(uint32_t *str1, uint32_t *str2)
{
    uint32_t number1[1]{ 0 };
    uint32_t number2[1]{ 0 };
   
    uint32_t res_number = 0;
    for (auto i = 0; i < 32; ++i)
    {
        atomicAdd(&number1[0], str1[i] * __powf(2, (31 - i)));
        atomicAdd(&number2[0], str2[i] * __powf(2, (31 - i)));
       // __syncthreads();
        /*result_xor[i] = str1[i] ^ str2[i];
        result_xor[i] = result_xor[i] ^ str3[i];*/
    }
    res_number = ((number1[0]) + (number2[0])) % 4294967296;
    //uint32_t number1 = 0;
    //uint32_t number2 = 0;
    //uint32_t res_number = 0;
    //// int i = threadIdx.x ;
    //for (int i = 0; i < 32; ++i)
    //    // if(i<32)
    //     //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //{
    //    uint32_t number3 = 1;
    //    for (auto j = 0; j < (31 - i); ++j)
    //    {
    //        number3 *= 2;
    //    }
    //    number1 += str1[i] * number3;
    //}
    //for (int i = 0; i < 32; ++i)
    //{
    //    uint32_t number3 = 1;
    //    for (auto j = 0; j < (31 - i); ++j)
    //    {
    //        number3 *= 2;
    //    }
    //    number2 += str2[i] * number3;
    //}
    //res_number = (number1 + number2) % 4294967296;
    //__syncthreads();
    return res_number;
    // delete[]result_sum;
     //delete[]result_sum;
}


__device__ uint32_t* xor_strs(uint32_t* str1, uint32_t* str2, unsigned int length, uint32_t* result_xor, uint32_t* str3)
{
   
    //uint32_t result_xor[32]{ 0 };


    for (auto i = 0; i <length; ++i)   
    {
        atomicXor(&(result_xor[i]),str1[i]);
        atomicXor(&(result_xor[i]), str2[i]);
        atomicXor(&(result_xor[i]), str3[i]);
        __syncthreads();
       /* result_xor[i] = str1[i] ^ str2[i];
        result_xor[i] = result_xor[i] ^ str3[i];*/
    }
    
   
   return result_xor;
   // delete[] result_xor;
}

__device__ uint32_t* rigth_rotate(uint32_t* str, unsigned int num,uint32_t* result_right_rotate)
{
    uint32_t number = 0;
   // uint32_t rotated_str[32]{ 0 };
    
   // memcpy(rotated_str, str, 4 * 32);
    number = str_to_32bitnumber(str);
   
    
    /*while (number != 0)
    {

        result_right_rotate[count] = number % 2;

        number /= 2;
        count--;

    }*/
   // printf("%u\n", number);
    number = ((number >> num) | (number << (32 - num)));
   // printf("%u\n", number);
    to_binary_32bit(number, result_right_rotate);
    return result_right_rotate;
    //return to_binary_32bit((number >> num | number << (32 - num)));

}

__device__ uint32_t* rigth_shift(uint32_t* str, unsigned int num, uint32_t* result_right_shift)
{

    //uint32_t shifted_str[32]{ 0 };
    
  //  memcpy(shifted_str, str, 4 * 32);
   
    uint32_t number = str_to_32bitnumber(str);
    /*while (number != 0)
    {

        result_right_shift[count] = number % 2;

        number /= 2;
        count--;

    }*/
    to_binary_32bit(number >> num, result_right_shift);
    return result_right_shift;

}



__device__ void password_xor_with_IPAD(char* password,unsigned int length, uint32_t* output_str)
{
    uint32_t binary_str[512]{ 0 };
    
    uint32_t IPAD[] = { 0,0,1,1,0,1,1,0 };
    
    int i = threadIdx.x+blockIdx.x*blockDim.x;
    
    output_str[i] = binary_str[i] ^ IPAD[i % 8];
     __syncthreads();
  
  
    
}

__device__ void password_xor_with_OPAD(char* password, unsigned int length, uint32_t *output_str)
{
    uint32_t binary_str[512]{ 0 };
    uint32_t OPAD [] = { 0,1,0,1,1,1,0,0 };
   
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    output_str[i] = binary_str[i] ^ OPAD[i % 8];
    __syncthreads();
   
}

 __device__ void preparation_sha256_with_IPAD(uint32_t* password_xor_with_ipad, uint32_t*prev_hash, uint32_t *output_str)
{
     uint32_t message[1024]{ 0 };
     //uint32_t prev_hash_hmac[256]{ 0 };
     for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < 512; i += gridDim.x * blockDim.x)
     {
         //  __syncthreads();
         message[i] = password_xor_with_ipad[i];
         __syncthreads();
     }

     for (int j = threadIdx.x + blockIdx.x * blockDim.x; j < 256; j += gridDim.x * blockDim.x)
     {

         message[j + 512] = prev_hash[j];
         __syncthreads();
     }
     // __syncthreads();
     message[768] = 1;
     message[1014] = 1;
     message[1015] = 1;
     for (int j = threadIdx.x + blockIdx.x * blockDim.x; j < 1024; j += gridDim.x * blockDim.x)
     {

         output_str[j] = message[j];
         __syncthreads();
     }
}

__device__ void preparation_sha256_with_OPAD(uint32_t* password_xor_with_opad, uint32_t* prev_hash, uint32_t* output_str)
{
    uint32_t message[1024]{ 0 };
    for (int s = threadIdx.x + blockIdx.x * blockDim.x; s < 512; s += gridDim.x * blockDim.x)
    {
        // __syncthreads();
        message[s] = password_xor_with_opad[s];
        __syncthreads();
    }

    for (int d = threadIdx.x + blockIdx.x * blockDim.x; d < 256; d += gridDim.x * blockDim.x)
    {
        //__syncthreads();
        message[d + 512] = prev_hash[d];
        __syncthreads();
    }

    message[768] = 1;
    message[1014] = 1;
    message[1015] = 1;
    for (int j = threadIdx.x + blockIdx.x * blockDim.x; j < 1024; j += gridDim.x * blockDim.x)
    {

        output_str[j] = message[j];
        __syncthreads();
    }
    
   
}

__device__ void main_loop_sha256(uint32_t* message, uint32_t* output_hash)
{

    uint32_t h0[32]{ 0, 1, 1, 0,  1, 0, 1, 0,  0, 0, 0, 0,  1, 0, 0, 1,  1, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 1 };
    uint32_t h1[32]{ 1, 0, 1, 1,  1, 0, 1, 1,  0, 1, 1, 0,  0, 1, 1, 1,  1, 0, 1, 0,  1, 1, 1, 0,  1, 0, 0, 0,  0, 1, 0, 1 };
    uint32_t h2[32]{ 0, 0, 1, 1,  1, 1, 0, 0,  0, 1, 1, 0,  1, 1, 1, 0,  1, 1, 1, 1,  0, 0, 1, 1,  0, 1, 1, 1,  0, 0, 1, 0 };
    uint32_t h3[32]{ 1, 0, 1, 0,  0, 1, 0, 1,  0, 1, 0, 0,  1, 1, 1, 1,  1, 1, 1, 1,  0, 1, 0, 1,  0, 0, 1, 1,  1, 0, 1, 0 };
    uint32_t h4[32]{ 0, 1, 0, 1,  0, 0, 0, 1,  0, 0, 0, 0,  1, 1, 1, 0,  0, 1, 0, 1,  0, 0, 1, 0,  0, 1, 1, 1,  1, 1, 1, 1 };
    uint32_t h5[32]{ 1, 0, 0, 1,  1, 0, 1, 1,  0, 0, 0, 0,  0, 1, 0, 1,  0, 1, 1, 0,  1, 0, 0, 0,  1, 0, 0, 0,  1, 1, 0, 0 };
    uint32_t h6[32]{ 0, 0, 0, 1,  1, 1, 1, 1,  1, 0, 0, 0,  0, 0, 1, 1,  1, 1, 0, 1,  1, 0, 0, 1,  1, 0, 1, 0,  1, 0, 1, 1 };
    uint32_t h7[32]{ 0, 1, 0, 1,  1, 0, 1, 1,  1, 1, 1, 0,  0, 0, 0, 0,  1, 1, 0, 0,  1, 1, 0, 1,  0, 0, 0, 1,  1, 0, 0, 1 };

    /*auto a = sum_strs_32bit(h0, h1);
    printf("a=%u\n", a);*/

    uint32_t part_message1[512]{ 0 };/*{ 0, 1, 1, 0, 1, 0, 0, 0, 0, 1, 1, 0, 0, 1, 0, 1, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 1, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 1, 1, 0, 1, 1, 0, 1, 1, 1, 1,
0,1,1,1,0,0,1,0, 0,1,1,0,1,1,0,0, 0,1,1,0,0,1,0,0, 1,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,1,0,1,1,0,0,0 };*/
    uint32_t part_message2[512]{ 0 };
   // memcpy(part_message1, message, 4 * 512);
   // memcpy(part_message2, message + 512, 4 * 512);

    int count = 1;//счётчик для 2ух итераций



    while (count < 3)
    {


        uint32_t S1[32]{ 0 };
        uint32_t ch[32]{ 0 };
        uint32_t temp1[32]{ 0 };
        uint32_t S0[32]{ 0 };
        uint32_t maj[32]{ 0 };
        uint32_t temp2[32]{ 0 };

        uint32_t a[32]{ 0 };
        uint32_t b[32]{ 0 };
        uint32_t c[32]{ 0 };
        uint32_t d[32]{ 0 };
        uint32_t e[32]{ 0 };
        uint32_t f[32]{ 0 };
        uint32_t g[32]{ 0 };
        uint32_t h[32]{ 0 };
        uint32_t extend_part_message1[64][32]{ 0 };
        



        if (count == 1)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message1 + (i * 32), 4 * 32);
            }
        }

        if (count == 2)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message2 + (i * 32), 4 * 32);
            }
        }
       /* printf("extend-message-before\n");
        for (auto i = 0; i < 64; ++i)
        {
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", extend_part_message1[i][j]);
            }
            printf("\n");
        }*/

        for (auto i = 16; i < 64; ++i)
        {
            for (int j = 0; j < 32; j++)
            {
                extend_part_message1[i][j] = extend_part_message1[i - 16][j] + extend_part_message1[i - 7][j] +
                    ((extend_part_message1[i - 15][(j + 25) % 32] + extend_part_message1[i - 15][(j + 14) % 32] + (j < 3 ? 0 : extend_part_message1[i - 15][(j + 29) % 32])) % 2) +
                    ((extend_part_message1[i - 2][(j + 15) % 32] + extend_part_message1[i - 2][(j + 13) % 32] + (j < 10 ? 0 : extend_part_message1[i - 2][(j + 22) % 32])) % 2);
            }
            for (int j = 31; j > 0; j--)
            {
                while (extend_part_message1[i][j] >= 2) {
                    extend_part_message1[i][j] -= 2;
                    extend_part_message1[i][j - 1]++;
                }
            }
            extend_part_message1[i][0] = extend_part_message1[i][0] % 2;
           /* uint32_t s0[32]{ 0 };
            uint32_t s1[32]{ 0 };
            uint32_t sum_ext1_2_s0_s1 = 0;
            uint32_t result_xor1[32]{ 0 };
            uint32_t result_xor2[32]{ 0 };
            uint32_t result_right_rotate1[32]{ 0 };
            uint32_t result_right_rotate2[32]{ 0 };
            uint32_t result_right_rotate3[32]{ 0 };
            uint32_t result_right_rotate4[32]{ 0 };
            uint32_t result_right_shift1[32]{ 0 };
            uint32_t result_right_shift2[32]{ 0 };

            rigth_rotate(extend_part_message1[i - 15], 7, result_right_rotate1);
            rigth_rotate(extend_part_message1[i - 15], 18, result_right_rotate2);

            rigth_shift(extend_part_message1[i - 15], 3, result_right_shift1);
            
            xor_strs(result_right_rotate1, result_right_rotate2, 32,
                result_xor1, result_right_shift1);
            memcpy(s0, result_xor1, 4 * 32);

           
            rigth_rotate(extend_part_message1[i - 2], 17, result_right_rotate3);
            rigth_rotate(extend_part_message1[i - 2], 19, result_right_rotate4);
            rigth_shift(extend_part_message1[i - 2], 10, result_right_shift2);

            xor_strs(result_right_rotate3, result_right_rotate4, 32,
                result_xor2, result_right_shift2);
            memcpy(s1, result_xor2, 4 * 32);

            


            sum_ext1_2_s0_s1 = (sum_strs_32bit(extend_part_message1[i - 16], s0) + sum_strs_32bit(extend_part_message1[i - 7], s1)) % 4294967296;

            to_binary_32bit(sum_ext1_2_s0_s1, extend_part_message1[i]);*/
          

        }
       
        /*printf("extend-message-after\n");
        for (auto i = 0; i < 64; ++i)
        {
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", extend_part_message1[i][j]);
            }
            printf("\n");
        }*/
        memcpy(a, h0, 4 * 32);
        memcpy(b, h1, 4 * 32);
        memcpy(c, h2, 4 * 32);
        memcpy(d, h3, 4 * 32);
        memcpy(e, h4, 4 * 32);
        memcpy(f, h5, 4 * 32);
        memcpy(g, h6, 4 * 32);
        memcpy(h, h7, 4 * 32);

        for (auto i = 0; i < 64; ++i)
        {
            /*uint32_t res_and1[32]{ 0 };
            uint32_t res_and2[32]{ 0 };
            uint32_t res_and3[32]{ 0 };
            uint32_t res_and4[32]{ 0 };
            uint32_t res_and5[32]{ 0 };

            uint32_t res_inv[32]{ 0 };
            uint32_t res_xor1[32]{ 0 };
            uint32_t res_xor2[32]{ 0 };
            uint32_t temp_res_xor[32]{ 0 };
            uint32_t res_xor3[32]{ 0 };
            uint32_t res_xor4[32]{ 0 };

            uint32_t res_right_rotate1[32]{ 0 };
            uint32_t res_right_rotate2[32]{ 0 };
            uint32_t res_right_rotate3[32]{ 0 };
            uint32_t res_right_rotate4[32]{ 0 };
            uint32_t res_right_rotate5[32]{ 0 };
            uint32_t res_right_rotate6[32]{ 0 };*/

           /* rigth_rotate(e, 6, res_right_rotate1);
            rigth_rotate(e, 11, res_right_rotate2);
            rigth_rotate(e, 25, res_right_rotate3);
            xor_strs(res_right_rotate1, res_right_rotate2, 32, res_xor1, res_right_rotate3);
            memcpy(S1, res_xor1, 4 * 32);*/


            /*memcpy(S1, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(e, 6)), to_binary_32bit(rigth_rotate(e, 11)), 32),
                to_binary_32bit(rigth_rotate(e, 25)), 32), 4 * 32);*/
            /*and_strs_32bit(e, f, res_and1);
            inverse_str_32bit(e, res_inv);
            and_strs_32bit(res_inv, g, res_and2);
            xor_strs(res_and1, res_and2, 32, res_xor2,temp_res_xor);
            memcpy(ch, res_xor2, 4 * 32);

            uint32_t to_binary_32bit0[32]{ 0 };*/
            //memcpy(ch, xor_strs(and_strs_32bit(e, f), and_strs_32bit(inverse_str_32bit(e), g), 32), 4 * 32);
           /* to_binary_32bit(((sum_strs_32bit(h, S1) + sum_strs_32bit(ch, extend_part_message1[i]) + round_consts[i]) % 4294967296), to_binary_32bit0);
            memcpy(temp1, to_binary_32bit0,4*32);
            rigth_rotate(a, 2, res_right_rotate4);
            rigth_rotate(a, 13, res_right_rotate5);
            rigth_rotate(a, 22, res_right_rotate6);
            xor_strs(res_right_rotate4,res_right_rotate5 , 32, res_xor3, res_right_rotate6);
            memcpy(S0, res_xor3, 4 * 32);*/

           // memcpy(S0, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(a, 2)), to_binary_32bit(rigth_rotate(a, 13)), 32),
           //     to_binary_32bit(rigth_rotate(a, 22)), 32), 4 * 32);

           // memcpy(maj, xor_strs(xor_strs(and_strs_32bit(a, b), and_strs_32bit(a, c), 32),
            //    and_strs_32bit(b, c), 32), 4 * 32);
            /*and_strs_32bit(a, b, res_and3);
            and_strs_32bit(a, c, res_and4);
            and_strs_32bit(b, c, res_and5);
            xor_strs(res_and3, res_and4, 32, res_xor4, res_and5);
            memcpy(maj, res_xor4, 4 * 32);*/
            uint32_t round_cnst[32]{ 0 };
            to_binary_32bit(round_consts[i], round_cnst);
           
            for (int j = 0; j < 32; j++)
            {
                temp2[j] = ((a[(j + 30) % 32] + a[(j + 19) % 32] + a[(j + 10) % 32]) % 2) +
                    (((b[j] == 1 && a[j] == 1 ? 1 : 0) + (c[j] == 1 && a[j] == 1 ? 1 : 0) + (c[j] == 1 && b[j] == 1 ? 1 : 0)) % 2);

                temp1[j] = h[j] +
                    ((e[(j + 26) % 32] + e[(j + 21) % 32] + e[(j + 7) % 32]) % 2) +
                    (((f[j] == 1 && e[j] == 1 ? 1 : 0) + (e[j] == 0 && g[j] == 1 ? 1 : 0)) % 2) +
                    extend_part_message1[i][j] +
                    round_cnst[j];
            }

            for (int j = 31; j > 0; j--)
            {
                while (temp2[j] >= 2) {
                    temp2[j] -= 2;
                    temp2[j - 1]++;
                }

                while (temp1[j] >= 2) {
                    temp1[j] -= 2;
                    temp1[j - 1]++;
                }
            }
            temp2[0] = temp2[0] % 2;
            temp1[0] = temp1[0] % 2;


            uint32_t to_binary_32bit1[32]{ 0 };
            uint32_t to_binary_32bit2[32]{ 0 };
            uint32_t to_binary_32bit3[32]{ 0 };
           

           /* to_binary_32bit(sum_strs_32bit(S0, maj), to_binary_32bit1);
            memcpy(temp2, to_binary_32bit1, 4 * 32);*/
            

            memcpy(h, g, 4 * 32);
            /*__syncthreads();
            printf("h\n");
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", h[j]);
            }
            printf("\n");*/

            memcpy(g, f, 4 * 32);
            /*__syncthreads();

            printf("g\n");
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", g[j]);
            }
            printf("\n");*/


            memcpy(f, e, 4 * 32);
           /* __syncthreads();

            printf("f\n");
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", f[j]);
            }
            printf("\n");*/

            to_binary_32bit(sum_strs_32bit(d, temp1), to_binary_32bit2);
            memcpy(e, to_binary_32bit2, 4 * 32);
            /*printf("e\n");
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", e[j]);
            }
            printf("\n");*/

            memcpy(d, c, 4 * 32);
            /*__syncthreads();
            printf("d\n");
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", d[j]);
            }
            printf("\n");*/
            memcpy(c, b, 4 * 32);
            /*__syncthreads();
            printf("c\n");
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", c[j]);
            }
            printf("\n");*/
            memcpy(b, a, 4 * 32);
           /* __syncthreads();
            printf("b\n");
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", b[j]);
            }
            printf("\n");*/

            to_binary_32bit(sum_strs_32bit(temp1, temp2),to_binary_32bit3);
            memcpy(a, to_binary_32bit3, 4 * 32);
            /*__syncthreads();
            printf("a\n");
            for (auto j = 0; j < 32; ++j)
            {
                printf("%u", a[j]);
            }
            printf("\n");*/
            
            
           

        }
         
        count++;

        to_binary_32bit(sum_strs_32bit(h0, a), h0);
        to_binary_32bit(sum_strs_32bit(h1, b), h1);
        to_binary_32bit(sum_strs_32bit(h2, c), h2);
        to_binary_32bit(sum_strs_32bit(h3, d), h3);
        to_binary_32bit(sum_strs_32bit(h4, e), h4);
        to_binary_32bit(sum_strs_32bit(h5, f), h5);
        to_binary_32bit(sum_strs_32bit(h6, g), h6);
        to_binary_32bit(sum_strs_32bit(h7, h), h7);

       
    }

    uint32_t* hash = new uint32_t[256]{ 0 };
    memcpy(hash, h0, 4 * 32);
    memcpy(hash + 32, h1, 4 * 32);
    memcpy(hash + 64, h2, 4 * 32);
    memcpy(hash + 96, h3, 4 * 32);
    memcpy(hash + 128, h4, 4 * 32);
    memcpy(hash + 160, h5, 4 * 32);
    memcpy(hash + 192, h6, 4 * 32);
    memcpy(hash + 224, h7, 4 * 32);
    memcpy(output_hash, hash, 4 * 256);

}




__device__ void hmac_sha256(int c, uint32_t* pass_xor_ipad, uint32_t* pass_xor_opad, uint32_t* prev_hash1)
{
    
    //for (auto i = 0; i < 1; ++i)
    //{
    //   //printf("%d\n", i);
    //    preparation_sha256_with_IPAD(pass_xor_ipad, prev_hash1, message);
    //   // 
    //    preparation_sha256_with_OPAD(pass_xor_opad, prev_hash1, message);
    //   
    //   // memcpy(prev_hash_hmac, xor_strs_256bit(prev_hash_hmac, prev_hash1), 4 * 256);
    //}
   
    //return message;
}

__global__ void password_xor_with_ipad_opad(char* password, unsigned int length, uint32_t* pass_xor_ipad, uint32_t* pass_xor_opad)
{
    password_xor_with_IPAD(password, length, pass_xor_ipad);
    password_xor_with_OPAD(password, length, pass_xor_opad);
}

__global__ void pbkdf2_hmac_sha256(unsigned int c, uint32_t* pass_xor_ipad, uint32_t* pass_xor_opad,uint32_t* salt, uint32_t* pbkdf2_hash)
{
   // int i = threadIdx.x;
    //preparation_sha256_with_IPAD(pass_xor_ipad, salt, pbkdf2_hash);
    uint32_t message[1024]{ 0 };
    uint32_t prev_hash_hmac[256]{ 0 };
    memcpy(prev_hash_hmac, salt, sizeof(uint32_t) * 256);
   // int i = 0;
    //memcpy(pbkdf2_hash, hmac_sha256(c, pass_xor_ipad, pass_xor_opad, salt), 4 * 256);
    for (auto j = 0; j < c; ++j)
    {
        printf("j=%d\n", j);

        preparation_sha256_with_IPAD(pass_xor_ipad, prev_hash_hmac, message);
        main_loop_sha256(message, prev_hash_hmac);
        preparation_sha256_with_OPAD(pass_xor_opad, prev_hash_hmac, message);
        main_loop_sha256(message, pbkdf2_hash);

        /* for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < 256; i += gridDim.x * blockDim.x)
         {
             pbkdf2_hash[i] = prev_hash_hmac[i] ^ pbkdf2_hash[i];
             __syncthreads();
         }*/
    }

}


//__device__ void pbkdf2_hmac_sha256()
//{
//    // 
//    // uint32_t *hash = new uint32_t[256]{ 0 };
//    // uint32_t *salt = new uint32_t[256]{ 0 };
//    // uint32_t *prev_hash = new uint32_t[256]{ 0 };
//    // //uint32_t* temp = new uint32_t[256]{ 0 };
//
//    // uint32_t password_xor_with_ipad[512]{ 0 };
//    // uint32_t password_xor_with_opad[512]{ 0 };
//
//    // memcpy(password_xor_with_ipad, password_xor_with_IPAD(password, length), 4 * 512);
//    // memcpy(password_xor_with_opad, password_xor_with_OPAD(password, length), 4 * 512);
//    //// int index = threadIdx.x + blockIdx.x*blockDim.x;
//    // uint32_t dklen = 256;
//    // uint32_t len = dklen / 256;
//    // uint32_t r = dklen - (len - 1) * 256;
//    // int index = threadIdx.x;
//    //
//    // //for (auto index = 0; index < len; ++index)
//    // while (index < len)
//    // {
//    //     salt[255] = index;
//    //     memcpy(prev_hash, salt, 4 * 256);
//    //     uint32_t temp_hash[256]{ 0 };
//    //     for (auto j = 0; j < c; ++j)
//    //     {
//    //         memcpy(prev_hash, hmac_sha256(prev_hash, password_xor_with_ipad, password_xor_with_opad), 4 * 256);
//    //         memcpy(temp_hash, xor_strs(temp_hash, prev_hash,256), 4 * 256);
//    //     }
//    //    
//    //     memcpy(hash + index * 256, temp_hash, 4 * 256);
//    // }
//    // delete[] salt;
//    // delete[] prev_hash;
//    // delete[] password_xor_with_ipad;
//    // delete[] password_xor_with_opad;
//    // return hash;
//
//}

uint32_t* random_salt(size_t Nbits)
{
    std::random_device rd;  //Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> int1(0, 1);
    uint32_t* str = new uint32_t[Nbits];
    //str.reserve(Nbits);
    for (size_t i = 0; i < Nbits; i++)
    {
        str[i] = int1(gen) ? 1 : 0;
    }
    return str;
};



int main()
{

    //std::string password = "1234";
    unsigned int len_hash_pbkdf2 = 1;
    //    uint32_t* pbkdf2_hash = new uint32_t[len_hash_pbkdf2 * 256];
    uint32_t* salt = new uint32_t[256]{ 0 };
    //memcpy(salt, random_salt(254), 4 * 254);



    uint32_t* password_xor_with_ipad = new uint32_t[512]{ 0 };
    uint32_t* password_xor_with_opad = new uint32_t[512]{ 0 };
    uint32_t* pbkdf2_hash = new uint32_t[len_hash_pbkdf2 * 1024]{ 0 };
    uint32_t* dev_password_xor_with_ipad;
    uint32_t* dev_password_xor_with_opad;
    uint32_t* dev_pbkdf2_hash;
    uint32_t* dev_salt;


    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_password_xor_with_ipad, 512 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }
    cudaStatus = hipMemcpy(dev_password_xor_with_ipad, password_xor_with_ipad, 512 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_password_xor_with_opad, 512 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }
    cudaStatus = hipMemcpy(dev_password_xor_with_opad, password_xor_with_opad, 512 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    password_xor_with_ipad_opad << <16, 32>> > ("1234", 4, dev_password_xor_with_ipad, dev_password_xor_with_opad);



    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(password_xor_with_ipad, dev_password_xor_with_ipad, 512 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(password_xor_with_opad, dev_password_xor_with_opad, 512 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << password_xor_with_ipad[i];
    }
    std::cout << std::endl;

    for (auto i = 0; i < 512; ++i)
    {
        std::cout << password_xor_with_opad[i];
    }
    std::cout << std::endl;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    //---------------------------------------------------------------------------------------------------------
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pbkdf2_hash, len_hash_pbkdf2 * 1024 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_pbkdf2_hash, pbkdf2_hash, len_hash_pbkdf2 * 1024 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_salt, 256 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_salt, salt, 256 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_password_xor_with_ipad, 512 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_password_xor_with_ipad, password_xor_with_ipad, 512 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_password_xor_with_opad, 512 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_password_xor_with_opad, password_xor_with_opad, 512 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    pbkdf2_hmac_sha256 << <1, 1>> > (2000,  dev_password_xor_with_ipad, dev_password_xor_with_opad, dev_salt, dev_pbkdf2_hash);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(pbkdf2_hash, dev_pbkdf2_hash, len_hash_pbkdf2 * 1024 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //main_loop_sha256(message_for_sha256, hash);
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << pbkdf2_hash[i];
    }
    std::cout << std::endl;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;

        //------------------------------------------------------------------------------------------------------------


                // Copy output vector from GPU buffer to host memory.






        return 0;
    Error:
        hipFree(dev_password_xor_with_ipad);
        hipFree(dev_password_xor_with_opad);
        //hipFree(dev_pbkdf2_hash);
        //hipFree(dev_salt);
        /*hipFree(device_hash);
        hipFree(hash);
        hipFree(device_message_1024bits);

        hipFree(dev_b);*/
    }
}


// Helper function for using CUDA to add vectors in parallel.

