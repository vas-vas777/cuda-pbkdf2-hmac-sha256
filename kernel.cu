#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <random>
#include <string>
#include <bitset>
#include <iostream>


//#include <openssl/hmac.h>




__constant__ const uint32_t round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

//__device__ uint32_t pow_uint32_t(uint32_t a, uint32_t x)
//{
//    uint32_t res = 1;
//    for (auto i = 0; i < x; ++i)
//    {
//        res = a * res;
//    }
//    return res;
//}


__device__ uint32_t* to_binary_32bit(uint32_t number)
{
    uint32_t binary_str[32]{ 0 };
    
    int count = 31;
    while (number!=0)
    {
       
        binary_str[count] = number % 2;
        
        number /= 2;
        count--;
       
    }
    return binary_str;
   
}

__device__ uint32_t str_to_32bitnumber(uint32_t* str)
{
    uint32_t number1 = 0;
    for (auto i = 0; i < 32; ++i)
    {
        uint32_t number2 = 1;
        for (auto j = 0; j < (31 - i); ++j)
        {
            number2 *= 2;
        }
        number1 += str[i] * number2;
    }

    return number1;
}

__device__ uint32_t* and_strs_32bit(uint32_t* str1, uint32_t* str2)
{
    uint32_t result_str[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = str1[i] & str2[i];
    }
    return result_str;
    
}

__device__ uint32_t* inverse_str_32bit(uint32_t* str1)
{
    uint32_t result_str[32]{ 0 };
   
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = !str1[i];
    }
    return result_str;
}

__device__ uint32_t* inverse_str_256bit(uint32_t* str1)
{
    uint32_t result_str[256]{ 0 };
    for (auto i = 0; i < 256; ++i)
    {
        result_str[i] = !str1[i];
    }
    return result_str;
    
}

__device__ uint32_t sum_strs_32bit(uint32_t str1[], uint32_t str2[])
{
    uint32_t number1 = 0;
    uint32_t number2 = 0;
    uint32_t res_number = 0;
    // int i = threadIdx.x ;
    for (int i = 0; i < 32; ++i)
        // if(i<32)
         //int i = blockIdx.x * blockDim.x + threadIdx.x;
    {
        uint32_t number3 = 1;
        for (auto j = 0; j < (31 - i); ++j)
        {
            number3 *= 2;
        }
        number1 += str1[i] * number3;
    }
    for (int i = 0; i < 32; ++i)
    {
        uint32_t number3 = 1;
        for (auto j = 0; j < (31 - i); ++j)
        {
            number3 *= 2;
        }
        number2 += str2[i] * number3;
    }
    res_number = (number1 + number2) % 4294967296;
   // __syncthreads();
    return res_number;
    // delete[]result_sum;
     //delete[]result_sum;
}

__device__ uint32_t* xor_strs_32bit(uint32_t* str1, uint32_t* str2)
{
    uint32_t result_xor[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_xor[i] = str1[i] ^ str2[i];
    }
    return result_xor;
}

__device__ uint32_t* xor_strs_256bit(uint32_t* str1, uint32_t* str2)
{
    uint32_t result_xor[256]{ 0 };
    for (auto i = 0; i < 256; ++i)
    {
        result_xor[i] = str1[i] ^ str2[i];
    }
    return result_xor;
}

__device__ uint32_t rigth_rotate(uint32_t* str, unsigned int num)
{
    uint32_t number = 0;
    uint32_t rotated_str[32]{ 0 };
    memcpy(rotated_str, str, 4 * 32);
    number = str_to_32bitnumber(rotated_str);
    return (number >> num | number << (32 - num));
    
}

__device__ uint32_t rigth_shift(uint32_t* str, unsigned int num)
{
    
    uint32_t shifted_str[32]{ 0 };
   memcpy(shifted_str, str, 4 * 32);
    uint32_t number = str_to_32bitnumber(shifted_str);
    return number >> num;
       
}




__device__ void password_xor_with_IPAD(char* password,unsigned int length, uint32_t* output_str)
{
    uint32_t binary_str[512]{ 0 };
    
    uint32_t IPAD[] = { 0,0,1,1,0,1,1,0 };
    
    int i = threadIdx.x+blockIdx.x*blockDim.x;
    
    output_str[i] = binary_str[i] ^ IPAD[i % 8];
     __syncthreads();
  
  
    
}

__device__ void password_xor_with_OPAD(char* password, unsigned int length, uint32_t *output_str)
{
    uint32_t binary_str[512]{ 0 };
    uint32_t OPAD [] = { 0,1,0,1,1,1,0,0 };
   
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    output_str[i] = binary_str[i] ^ OPAD[i % 8];
    __syncthreads();
   
    
}

 __device__ void preparation_sha256_with_IPAD(uint32_t* password_xor_with_ipad, uint32_t*prev_hash, uint32_t *output_str)
{
     
     /*for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < 1024; i += gridDim.x * blockDim.x)
     {
         output_str[i] = binary_str[i];
     }
     __syncthreads();*/
    
    // __syncthreads();
     
   // memcpy_1024bits(output_str, binary_str);
    // memcpy(output_str, binary_str, 4 * 1024);
     //return output_str;
}

__device__ void preparation_sha256_with_OPAD(uint32_t* password_xor_with_opad, uint32_t* prev_hash, uint32_t* output_str)
{
    //uint32_t binary_str[1024]{ 0 };
    
    
   /*for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < 1024; i += gridDim.x * blockDim.x)
   {
       output_str[i] = binary_str[i];
   }
   __syncthreads();*/
    
    //memcpy_512bits(binary_str, password_xor_with_opad);
    //memcpy_256bits_prev_hash(binary_str, prev_hash);
    //// memcpy(binary_str, password_xor_with_opad, 4 * 512);
    //// memcpy(binary_str + 512, prev_hash, 4 * 256);
   
    //memcpy_1024bits(output_str, binary_str);

    
   
}

__device__ void main_loop_sha256(uint32_t* message, uint32_t* output_hash)
{
   
    uint32_t h0[32] { 0, 1, 1, 0,  1, 0, 1, 0,  0, 0, 0, 0,  1, 0, 0, 1,  1, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 1 };
    uint32_t h1[32] { 1, 0, 1, 1,  1, 0, 1, 1,  0, 1, 1, 0,  0, 1, 1, 1,  1, 0, 1, 0,  1, 1, 1, 0,  1, 0, 0, 0,  0, 1, 0, 1 };
    uint32_t h2[32] { 0, 0, 1, 1,  1, 1, 0, 0,  0, 1, 1, 0,  1, 1, 1, 0,  1, 1, 1, 1,  0, 0, 1, 1,  0, 1, 1, 1,  0, 0, 1, 0 };
    uint32_t h3[32] { 1, 0, 1, 0,  0, 1, 0, 1,  0, 1, 0, 0,  1, 1, 1, 1,  1, 1, 1, 1,  0, 1, 0, 1,  0, 0, 1, 1,  1, 0, 1, 0 };
    uint32_t h4[32] { 0, 1, 0, 1,  0, 0, 0, 1,  0, 0, 0, 0,  1, 1, 1, 0,  0, 1, 0, 1,  0, 0, 1, 0,  0, 1, 1, 1,  1, 1, 1, 1 };
    uint32_t h5[32] { 1, 0, 0, 1,  1, 0, 1, 1,  0, 0, 0, 0,  0, 1, 0, 1,  0, 1, 1, 0,  1, 0, 0, 0,  1, 0, 0, 0,  1, 1, 0, 0 };
    uint32_t h6[32] { 0, 0, 0, 1,  1, 1, 1, 1,  1, 0, 0, 0,  0, 0, 1, 1,  1, 1, 0, 1,  1, 0, 0, 1,  1, 0, 1, 0,  1, 0, 1, 1 };
    uint32_t h7[32] { 0, 1, 0, 1,  1, 0, 1, 1,  1, 1, 1, 0,  0, 0, 0, 0,  1, 1, 0, 0,  1, 1, 0, 1,  0, 0, 0, 1,  1, 0, 0, 1 };
    uint32_t part_message1[512]{ 0 };/*{ 0, 1, 1, 0, 1, 0, 0, 0, 0, 1, 1, 0, 0, 1, 0, 1, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 1, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 1, 1, 0, 1, 1, 0, 1, 1, 1, 1,
0,1,1,1,0,0,1,0, 0,1,1,0,1,1,0,0, 0,1,1,0,0,1,0,0, 1,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,1,0,1,1,0,0,0 };*/
    uint32_t part_message2[512]{ 0 };
    memcpy(part_message1, message, 4 * 512);
    memcpy(part_message2, message + 512, 4 * 512);
   
    int count = 1;//счётчик для 2ух итераций
   

   
    while (count < 3)
    {


        uint32_t S1[32]{ 0 };
        uint32_t ch[32]{ 0 };
        uint32_t temp1[32]{ 0 };
        uint32_t S0[32]{ 0 };
        uint32_t maj[32]{ 0 };
        uint32_t temp2[32]{ 0 };

        uint32_t a[32]{ 0 };
        uint32_t b[32]{ 0 };
        uint32_t c[32]{ 0 };
        uint32_t d[32]{ 0 };
        uint32_t e[32]{ 0 };
        uint32_t f[32]{ 0 };
        uint32_t g[32]{ 0 };
        uint32_t h[32]{ 0 };
        uint32_t extend_part_message1[64][32];
        /*for (auto i = 0; i < 64; ++i)
        {
            extend_part_message1[i] = new uint32_t[32]{ 0 };
        }*/

        if (count == 1)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message1 + (i * 32), 4 * 32);
                
            }
        }

        if (count == 2)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message2 + (i * 32), 4 * 32);
                
            }
        }

        /*printf("ext_message_before\n");
        for (auto i = 0; i < 64; ++i)
        {
            for (auto j = 0; j < 32; ++j)
            {
                printf("%d", extend_part_message1[i][j]);
            }
            printf("\n");
        }
        printf("\n");*/
        
        for (auto i = 16; i < 64; ++i)
        {
            uint32_t s0[32]{ 0 };
            uint32_t s1[32]{ 0 };
            uint32_t sum_ext1_2_s0_s1 = 0;


            memcpy(s0, xor_strs_32bit(xor_strs_32bit(to_binary_32bit(rigth_rotate(extend_part_message1[i - 15], 7)),
                to_binary_32bit(rigth_rotate(extend_part_message1[i - 15], 18))),
                to_binary_32bit(rigth_shift(extend_part_message1[i - 15], 3))), 4 * 32);



            memcpy(s1, xor_strs_32bit(xor_strs_32bit(to_binary_32bit(rigth_rotate(extend_part_message1[i - 2], 17)),
                to_binary_32bit(rigth_rotate(extend_part_message1[i - 2], 19))),
                to_binary_32bit(rigth_shift(extend_part_message1[i - 2], 10))), 4 * 32);


            sum_ext1_2_s0_s1 = (sum_strs_32bit(extend_part_message1[i - 16], s0) + sum_strs_32bit(extend_part_message1[i - 7], s1)) % 4294967296;


            memcpy(extend_part_message1[i], to_binary_32bit(sum_ext1_2_s0_s1), 4 * 32);

        }
        /*printf("ext_message_after\n");
        for (auto i = 0; i < 64; ++i)
        {
            for (auto j = 0; j < 32; ++j)
            {
                printf("%d", extend_part_message1[i][j]);
            }
            printf("\n");
        }
        printf("\n");*/

        memcpy(a, h0, 4 * 32);
        memcpy(b, h1, 4 * 32);
        memcpy(c, h2, 4 * 32);
        memcpy(d, h3, 4 * 32);
        memcpy(e, h4, 4 * 32);
        memcpy(f, h5, 4 * 32);
        memcpy(g, h6, 4 * 32);
        memcpy(h, h7, 4 * 32);
#pragma unroll 64
        for (auto i = 0; i < 64; ++i)
        {

            memcpy(S1, xor_strs_32bit(xor_strs_32bit(to_binary_32bit(rigth_rotate(e, 6)), to_binary_32bit(rigth_rotate(e, 11))),
                to_binary_32bit(rigth_rotate(e, 25))), 4 * 32);
            memcpy(ch, xor_strs_32bit(and_strs_32bit(e, f), and_strs_32bit(inverse_str_32bit(e), g)), 4 * 32);

            memcpy(temp1, to_binary_32bit((sum_strs_32bit(h, S1) + sum_strs_32bit(ch, extend_part_message1[i]) + round_consts[i]) % 4294967296), 4 * 32);

            memcpy(S0, xor_strs_32bit(xor_strs_32bit(to_binary_32bit(rigth_rotate(a, 2)), to_binary_32bit(rigth_rotate(a, 13))),
                to_binary_32bit(rigth_rotate(a, 22))), 4 * 32);

            memcpy(maj, xor_strs_32bit(xor_strs_32bit(and_strs_32bit(a, b), and_strs_32bit(a, c)),
                and_strs_32bit(b, c)), 4 * 32);

            memcpy(temp2, to_binary_32bit(sum_strs_32bit(S0, maj)), 4 * 32);

            memcpy(h, g, 4 * 32);

            memcpy(g, f, 4 * 32);

            memcpy(f, e, 4 * 32);
            memcpy(e, to_binary_32bit(sum_strs_32bit(d, temp1)), 4 * 32);

            memcpy(d, c, 4 * 32);
            memcpy(c, b, 4 * 32);
            memcpy(b, a, 4 * 32);

            memcpy(a, to_binary_32bit(sum_strs_32bit(temp1, temp2)), 4 * 32);
            // __syncthreads();
           //  __syncthreads();
        }
        count++;

        memcpy(h0, to_binary_32bit(sum_strs_32bit(h0, a)), 4 * 32);
        memcpy(h1, to_binary_32bit(sum_strs_32bit(h1, b)), 4 * 32);
        memcpy(h2, to_binary_32bit(sum_strs_32bit(h2, c)), 4 * 32);
        memcpy(h3, to_binary_32bit(sum_strs_32bit(h3, d)), 4 * 32);
        memcpy(h4, to_binary_32bit(sum_strs_32bit(h4, e)), 4 * 32);
        memcpy(h5, to_binary_32bit(sum_strs_32bit(h5, f)), 4 * 32);
        memcpy(h6, to_binary_32bit(sum_strs_32bit(h6, g)), 4 * 32);
        memcpy(h7, to_binary_32bit(sum_strs_32bit(h7, h)), 4 * 32);

    }
    
    uint32_t hash[256]{ 0 };
    memcpy(hash, h0, 4*32);
    memcpy(hash + 32, h1, 4*32);
    memcpy(hash + 64, h2, 4*32);
    memcpy(hash + 96, h3, 4*32);
    memcpy(hash + 128, h4, 4*32);
    memcpy(hash + 160, h5, 4*32);
    memcpy(hash + 192, h6, 4*32);
    memcpy(hash + 224, h7, 4*32);
    
    memcpy(output_hash, hash, 4 * 256);
  
}



__device__ void hmac_sha256(int c, uint32_t* pass_xor_ipad, uint32_t* pass_xor_opad, uint32_t* prev_hash1)
{
    
    //for (auto i = 0; i < 1; ++i)
    //{
    //   //printf("%d\n", i);
    //    preparation_sha256_with_IPAD(pass_xor_ipad, prev_hash1, message);
    //   // 
    //    preparation_sha256_with_OPAD(pass_xor_opad, prev_hash1, message);
    //   
    //   // memcpy(prev_hash_hmac, xor_strs_256bit(prev_hash_hmac, prev_hash1), 4 * 256);
    //}
   
    //return message;
}

__global__ void password_xor_with_ipad_opad(char* password, unsigned int length, uint32_t* pass_xor_ipad, uint32_t* pass_xor_opad)
{
    password_xor_with_IPAD(password, length, pass_xor_ipad);
    password_xor_with_OPAD(password, length, pass_xor_opad);
}

__global__ void pbkdf2_hmac_sha256(unsigned int c, uint32_t* pass_xor_ipad, uint32_t* pass_xor_opad,uint32_t* salt, uint32_t* pbkdf2_hash)
{
   // int i = threadIdx.x;
    //preparation_sha256_with_IPAD(pass_xor_ipad, salt, pbkdf2_hash);
    uint32_t message[1024]{ 0 };
    uint32_t prev_hash_hmac[256]{ 0 };
   // int i = 0;
    //memcpy(pbkdf2_hash, hmac_sha256(c, pass_xor_ipad, pass_xor_opad, salt), 4 * 256);
    for (auto j = 0; j < c; ++j)
    {
        //printf("%d\n", j);
        for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < 512; i += gridDim.x * blockDim.x)
        {
            //  __syncthreads();
            message[i] = pass_xor_ipad[i];
            __syncthreads();
        }

        for (int j = threadIdx.x + blockIdx.x * blockDim.x; j < 256; j += gridDim.x * blockDim.x)
        {

           message[j + 512] = salt[j];
            __syncthreads();
        }
        // __syncthreads();
        message[768] = 1;
        message[1014] = 1;
        message[1015] = 1;

        main_loop_sha256(message, salt);
        __syncthreads();

        for (int s = threadIdx.x + blockIdx.x * blockDim.x; s < 512; s += gridDim.x * blockDim.x)
        {
            // __syncthreads();
            message[s] = pass_xor_opad[s];
            __syncthreads();
        }

        for (int d = threadIdx.x + blockIdx.x * blockDim.x; d < 256; d += gridDim.x * blockDim.x)
        {
            //__syncthreads();
            message[d + 512] = salt[d];
            __syncthreads();
        }

        message[768] = 1;
        message[1014] = 1;
        message[1015] = 1;
        __syncthreads();

      
      
        main_loop_sha256(message, salt);
        __syncthreads();
      

        for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < 256; i += gridDim.x * blockDim.x)
        {
              __syncthreads();
            prev_hash_hmac[i] = prev_hash_hmac[i] ^ salt[i];
            __syncthreads();
        }
        __syncthreads();

        // __syncthreads();
    }
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < 256; i += gridDim.x * blockDim.x)
    {
        //  __syncthreads();
        pbkdf2_hash[i] = prev_hash_hmac[i];
        __syncthreads();
    }

}


//__device__ void pbkdf2_hmac_sha256()
//{
//    // 
//    // uint32_t *hash = new uint32_t[256]{ 0 };
//    // uint32_t *salt = new uint32_t[256]{ 0 };
//    // uint32_t *prev_hash = new uint32_t[256]{ 0 };
//    // //uint32_t* temp = new uint32_t[256]{ 0 };
//
//    // uint32_t password_xor_with_ipad[512]{ 0 };
//    // uint32_t password_xor_with_opad[512]{ 0 };
//
//    // memcpy(password_xor_with_ipad, password_xor_with_IPAD(password, length), 4 * 512);
//    // memcpy(password_xor_with_opad, password_xor_with_OPAD(password, length), 4 * 512);
//    //// int index = threadIdx.x + blockIdx.x*blockDim.x;
//    // uint32_t dklen = 256;
//    // uint32_t len = dklen / 256;
//    // uint32_t r = dklen - (len - 1) * 256;
//    // int index = threadIdx.x;
//    //
//    // //for (auto index = 0; index < len; ++index)
//    // while (index < len)
//    // {
//    //     salt[255] = index;
//    //     memcpy(prev_hash, salt, 4 * 256);
//    //     uint32_t temp_hash[256]{ 0 };
//    //     for (auto j = 0; j < c; ++j)
//    //     {
//    //         memcpy(prev_hash, hmac_sha256(prev_hash, password_xor_with_ipad, password_xor_with_opad), 4 * 256);
//    //         memcpy(temp_hash, xor_strs(temp_hash, prev_hash,256), 4 * 256);
//    //     }
//    //    
//    //     memcpy(hash + index * 256, temp_hash, 4 * 256);
//    // }
//    // delete[] salt;
//    // delete[] prev_hash;
//    // delete[] password_xor_with_ipad;
//    // delete[] password_xor_with_opad;
//    // return hash;
//
//}

uint32_t* random_salt(size_t Nbits)
{
    std::random_device rd;  //Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> int1(0, 1);
    uint32_t* str = new uint32_t[Nbits];
    //str.reserve(Nbits);
    for (size_t i = 0; i < Nbits; i++)
    {
        str[i] = int1(gen) ? 1 : 0;
    }
    return str;
};



int main()
{

    //std::string password = "1234";
    unsigned int len_hash_pbkdf2 = 1;
    //    uint32_t* pbkdf2_hash = new uint32_t[len_hash_pbkdf2 * 256];
    uint32_t* salt = new uint32_t[256]{ 0 };
    //memcpy(salt, random_salt(254), 4 * 254);



    uint32_t* password_xor_with_ipad = new uint32_t[512]{ 0 };
    uint32_t* password_xor_with_opad = new uint32_t[512]{ 0 };
    uint32_t* pbkdf2_hash = new uint32_t[len_hash_pbkdf2 * 1024]{ 0 };
    uint32_t* dev_password_xor_with_ipad;
    uint32_t* dev_password_xor_with_opad;
    uint32_t* dev_pbkdf2_hash;
    uint32_t* dev_salt;


    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_password_xor_with_ipad, 512 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }
    cudaStatus = hipMemcpy(dev_password_xor_with_ipad, password_xor_with_ipad, 512 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_password_xor_with_opad, 512 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }
    cudaStatus = hipMemcpy(dev_password_xor_with_opad, password_xor_with_opad, 512 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    password_xor_with_ipad_opad << <16, 32>> > ("1234", 4, dev_password_xor_with_ipad, dev_password_xor_with_opad);



    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(password_xor_with_ipad, dev_password_xor_with_ipad, 512 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(password_xor_with_opad, dev_password_xor_with_opad, 512 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << password_xor_with_ipad[i];
    }
    std::cout << std::endl;

    for (auto i = 0; i < 512; ++i)
    {
        std::cout << password_xor_with_opad[i];
    }
    std::cout << std::endl;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    //---------------------------------------------------------------------------------------------------------
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_pbkdf2_hash, len_hash_pbkdf2 * 1024 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_pbkdf2_hash, pbkdf2_hash, len_hash_pbkdf2 * 1024 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_salt, 256 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_salt, salt, 256 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_password_xor_with_ipad, 512 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_password_xor_with_ipad, password_xor_with_ipad, 512 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_password_xor_with_opad, 512 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_password_xor_with_opad, password_xor_with_opad, 512 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    pbkdf2_hmac_sha256 << <1, 1024 >> > (3,  dev_password_xor_with_ipad, dev_password_xor_with_opad, dev_salt, dev_pbkdf2_hash);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(pbkdf2_hash, dev_pbkdf2_hash, len_hash_pbkdf2 * 1024 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //main_loop_sha256(message_for_sha256, hash);
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << pbkdf2_hash[i];
    }
    std::cout << std::endl;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;

        //------------------------------------------------------------------------------------------------------------


                // Copy output vector from GPU buffer to host memory.






        return 0;
    Error:
        hipFree(dev_password_xor_with_ipad);
        hipFree(dev_password_xor_with_opad);
        //hipFree(dev_pbkdf2_hash);
        //hipFree(dev_salt);
        /*hipFree(device_hash);
        hipFree(hash);
        hipFree(device_message_1024bits);

        hipFree(dev_b);*/
    }
}


// Helper function for using CUDA to add vectors in parallel.

