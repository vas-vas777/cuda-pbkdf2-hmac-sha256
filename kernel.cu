#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <bitset>
#include <iostream>


const __int64 round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
                            0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
                            0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
                            0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
                            0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
                            0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
                            0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
                            0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };


hipError_t addWithCuda(char* binary_pass, int length_block);

__host__ __int64* to_binary_32bit(__int64 number)
{
    __int64* binary_str = new __int64[32]{ 0 };
    int count = 31;
    //std::cout << number << std::endl;
    while (number!=0)
    {
        //std::cout << count << std::endl;
        binary_str[count] = number % 2;
        number /= 2;
        count--;
    }
    /*for (auto i = 0; i < 32; ++i)
    {
        std::cout << binary_str[i];
    }*/
    return binary_str;
    //delete[] binary_str;
}

__host__ __int64* and_strs_32bit(__int64* str1, __int64* str2)
{
    __int64* result_str = new __int64[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = str1[i] & str2[i];
    }
    return result_str;
    //delete[] result_str;
}

__host__ __int64* inverse_str_32bit(__int64* str1)
{
    __int64* result_str = new __int64[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = ~str1[i];
    }
    return result_str;
    //delete[] result_str;
}

__host__ __int64 sum_strs_32bit(__int64* str1, __int64* str2)
{
    //__int64* result_sum = new __int64[32]{ 0 };
    __int64 number1 = 0;
    __int64 number2 = 0;
    for (auto i = 0; i < 32; ++i)
    {
        number1 += str1[i] * pow(2, (31 - i));
    }
    for (auto i = 0; i < 32; ++i)
    {
        number2 += str2[i] * pow(2, (31 - i));
    }
    //std::cout << number1 << std::endl;
    //std::cout << number2 << std::endl;
    number2 = (number1 + number2) % 4294967296;

    //std::cout << number2 << std::endl;
    return number2;
   // delete[]result_sum;
    //delete[]result_sum;
}

__host__ __int64* xor_strs_32bit(__int64* str1, __int64* str2)
{
    __int64* result_xor = new __int64[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_xor[i] = str1[i] ^ str2[i];
    }
    return result_xor;
    //delete[]result_xor;
}

__host__ __int64* rigth_rotate(__int64* str,unsigned int num)
{
    __int64 last = 0;
    __int64* rotated_str = str;
    for (unsigned int count = 1; count <= num; ++count)
    {
        last = rotated_str[31];
        for (auto i = 31; i > 0; i--)
        {
            rotated_str[i] = rotated_str[i - 1];
        }
        rotated_str[0] = last;
       /* std::cout <<count<< std::endl;
        for (auto i = 0; i < 32; ++i)
        {
            std::cout << rotated_str[i];
        }
        std::cout << std::endl;*/
    }
    
    return rotated_str;
    //delete[] rotated_str;
}

__host__ __int64* rigth_shift(__int64* str, unsigned int num)
{
    //__int64 last = 0;
   // for (auto count = 0; count < num; ++count)
   // {
       // last = str[31];
    __int64* shifted_str = str;
        for (unsigned int i = 31; i > 0; --i)
        {
            if (i >= num)
            {
                shifted_str[i] = shifted_str[i - num];
            }
            else
            {
                shifted_str[i] = 0;
            }
        }
       // str[0] = last;
  //  }
    /*for (auto i = 0; i < 32; ++i)
    {
        std::cout << shifted_str[i];
    }*/
    return shifted_str;
    //delete[] shifted_str;
}



__host__ __int64* password_xor_with_IPAD(char* password,int length)
{
    __int64* binary_str = new __int64[512]{ 0 };
    __int64 IPAD[] = { 0,0,1,1,0,1,1,0 };
    __int64* ipad = new __int64[512]{ 0 };
    for (auto i = 0; i < 512; ++i)
    {
        if (i < 8 * length)
        {
            binary_str[i] = (0 != (password[i / 8] & 1 << (~i & 7)));
        }
        else
        {
            binary_str[i] = 0;
        }
    }
    for (auto i = 0; i < 512; ++i)
    {
        ipad[i] = IPAD[i % 8];
        
    }
   /* for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << ipad[i];
    }
    std::cout << std::endl;*/
    for (auto i = 0; i < 512; ++i)
    {
        binary_str[i] = binary_str[i] ^ ipad[i];
    }
    std::cout << "password_xor_with_IPAD 512 bits" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    //std::cout << sizeof(binary_str) << std::endl;
    
    return binary_str;
   /* delete[] binary_str;
    delete[] ipad;*/
    //std::string binary_str = {};
    //std::string IPAD = {};
    ////std::cout << binary_pass << std::endl;
    //for (auto i = 0; i < 64; ++i) //перевод в двоичную строку пароля
    //{
    //    if (i < password.size())
    //    {
    //        binary_str.append(std::bitset<8>(password.c_str()[i]).to_string());
    //        IPAD.append(std::bitset<8>('0x36').to_string());
    //    }
    //    else
    //    {
    //        binary_str.append(std::bitset<8>(0).to_string());
    //        IPAD.append(std::bitset<8>('0x36').to_string());
    //    }
    //}
    /////std::cout << binary_str << std::endl;
    ////std::cout << std::endl;
    ////std::cout << IPAD << std::endl;
    ////std::cout << binary_str.size() << " " << IPAD.size() << std::endl;
    //binary_str = (std::bitset<512>(binary_str) ^ std::bitset<512>(IPAD)).to_string();
    //return binary_str;
    //std::cout << binary_str << std::endl;
    
}

__host__ __int64* password_xor_with_OPAD(char* password, int length)
{
    __int64* binary_str = new __int64[512]{ 0 };
    __int64 OPAD [] = { 0,1,0,1,1,1,0,0 };
    __int64* opad = new __int64[512]{ 0 };
    for (auto i = 0; i < 512; ++i)
    {
        if (i < 8 * length)
        {
            binary_str[i] = (0 != (password[i / 8] & 1 << (~i & 7)));
        }
        else
        {
            binary_str[i] = 0;
        }
    }
    for (auto i = 0; i < 512; ++i)
    {
        opad[i] = OPAD[i % 8];

    }
    /*for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << opad[i];
    }
    std::cout << std::endl;*/
    for (auto i = 0; i < 512; ++i)
    {
        binary_str[i] = binary_str[i] ^ opad[i];
    }
   /* for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }*/

    
    return binary_str;
    delete[] binary_str;
    delete[] opad;
    //std::string binary_str = {};
    //std::string IPAD = {};
    ////std::cout << binary_pass << std::endl;
    //for (auto i = 0; i < 64; ++i) //перевод в двоичную строку пароля
    //{
    //    if (i < password.size())
    //    {
    //        binary_str.append(std::bitset<8>(password.c_str()[i]).to_string());
    //        IPAD.append(std::bitset<8>('0x5C').to_string());
    //    }
    //    else
    //    {
    //        binary_str.append(std::bitset<8>(0).to_string());
    //        IPAD.append(std::bitset<8>('0x5C').to_string());
    //    }
    //}
    /////std::cout << binary_str << std::endl;
    ////std::cout << std::endl;
    ////std::cout << IPAD << std::endl;
    ////std::cout << binary_str.size() << " " << IPAD.size() << std::endl;
    //binary_str = (std::bitset<512>(binary_str) ^ std::bitset<512>(IPAD)).to_string();
    //return binary_str;
}

 __host__ __int64* preparation_sha256_with_IPAD(__int64* password_xor_with_ipad, __int64*prev_hash)
{
     __int64* binary_str = new __int64[1024]{ 0 };
     memcpy(binary_str, password_xor_with_ipad, 8*512);
    memcpy(binary_str + 512, prev_hash, 8 * 256);
    binary_str[768] = 1;
    binary_str[1014] = 1;
    binary_str[1015] = 1;
    //memmove(binary_str, password_xor_with_ipad, 512);
    std::cout << "preparation_sha256_with_IPAD-1024 bits" << std::endl;
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    //memcpy(output_message, binary_str, 8 * 1024);//копируем все 1024 бита сообщения
   
    //for (auto i = 0; i < 2048; ++i)
    //{
    //    std::cout << output_message[i];
    //}
    //std::cout << std::endl;
    
    return binary_str;
    
}

__host__ __int64* preparation_sha256_with_OPAD(__int64* password_xor_with_opad, __int64* prev_hash)
{
    __int64* binary_str = new __int64[1024]{ 0 };
   // __int64* output_message = new __int64[2048]{ 0 };
    //memcpy(binary_str, password_xor_with_ipad, 512);
    /*std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << password_xor_with_opad[i];
    }
    std::cout << std::endl;*/

    memcpy(binary_str, password_xor_with_opad, 8 * 512);
    memcpy(binary_str + 512, prev_hash, 8 * 256);
    binary_str[768] = 1;
    binary_str[1014] = 1;
    binary_str[1015] = 1;
   
    return binary_str;
   
}

__host__ __int64* main_loop_sha256_with_ipad(__int64* message)
{
    __int64* h0 = new __int64[] { 0, 1, 1, 0,  1, 0, 1, 0,  0, 0, 0, 0,  1, 0, 0, 1,  1, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 1 };
    __int64* h1 = new __int64[] { 1, 0, 1, 1,  1, 0, 1, 1,  0, 1, 1, 0,  0, 1, 1, 1,  1, 0, 1, 0,  1, 1, 1, 0,  1, 0, 0, 0,  0, 1, 0, 1 };
    __int64* h2 = new __int64[] { 0, 0, 1, 1,  1, 1, 0, 0,  0, 1, 1, 0,  1, 1, 1, 0,  1, 1, 1, 1,  0, 0, 1, 1,  0, 1, 1, 1,  0, 0, 1, 0 };
    __int64* h3 = new __int64[] { 1, 0, 1, 0,  0, 1, 0, 1,  0, 1, 0, 0,  1, 1, 1, 1,  1, 1, 1, 1,  0, 1, 0, 1,  0, 0, 1, 1,  1, 0, 1, 0 };
    __int64* h4 = new __int64[] { 0, 1, 0, 1,  0, 0, 0, 1,  0, 0, 0, 0,  1, 1, 1, 0,  0, 1, 0, 1,  0, 0, 1, 0,  0, 1, 1, 1,  1, 1, 1, 1 };
    __int64* h5 = new __int64[] { 1, 0, 0, 1,  1, 0, 1, 1,  0, 0, 0, 0,  0, 1, 0, 1,  0, 1, 1, 0,  1, 0, 0, 0,  1, 0, 0, 0,  1, 1, 0, 0 };
    __int64* h6 = new __int64[] { 0, 0, 0, 1,  1, 1, 1, 1,  1, 0, 0, 0,  0, 0, 1, 1,  1, 1, 0, 1,  1, 0, 0, 1,  1, 0, 1, 0,  1, 0, 1, 1 };
    __int64* h7 = new __int64[] { 0, 1, 0, 1,  1, 0, 1, 1,  1, 1, 1, 0,  0, 0, 0, 0,  1, 1, 0, 0,  1, 1, 0, 1,  0, 0, 0, 1,  1, 0, 0, 1 };
    __int64* part_message1 = new __int64[512]{ 0 };
    __int64* part_message2 = new __int64[512]{ 0 };
    memcpy(part_message1, message, 8 * 512);
    memcpy(part_message2, message + 512, 8 * 512);
    std::cout << "message" << std::endl;
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << message[i];
    }
    std::cout << std::endl;
    std::cout << "part_message1" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << part_message1[i];
    }
    std::cout << std::endl;
    std::cout << "part_message2" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << part_message2[i];
    }
    std::cout << std::endl;
    int count = 1;//счётчик для 2ух итераций
    __int64* s0 = new __int64[32]{ 0 };
    __int64* s1 = new __int64[32]{ 0 };
    __int64* S1 = new __int64[32]{ 0 };
    __int64* ch = new __int64[32]{ 0 };
    __int64* temp1 = new __int64[32]{ 0 };
    __int64* S0 = new __int64[32]{ 0 };
    __int64* maj = new __int64[32]{ 0 };
    __int64* temp2 = new __int64[32]{ 0 };

    __int64 sum_ext1_2_s0_s1 = 0;
    //__int64 sum_ext2_s1 = 0;

    __int64* a = 0;
    __int64* b = 0;
    __int64* c = 0;
    __int64* d = 0;
    __int64* e = 0;
    __int64* f = 0;
    __int64* g = 0;
    __int64* h = 0;
   // std::cout << "h0" << std::endl;
   // for (auto i = 0; i < 32; ++i)
   // {
   //     std::cout << h0[i];
   // }
   // std::cout << std::endl;
   ///* std::cout << "h1" << std::endl;
   // for (auto i = 0; i < 32; ++i)
   // {
   //     std::cout << h1[i];
   // }
   // std::cout << std::endl;*/
   // s0 = rigth_shift(h0, 7);
   // for (auto i = 0; i < 32; ++i)
   // {
   //     std::cout << s0[i];
   // }
    /*std::cout << "a" << std::endl;
    
    std::cout << "h1" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h1[i];
    }
    s1 = rigth_rotate(h1, 2);
    std::cout << "b" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << s1[i];
    }
    ch=xor_strs_32bit(rigth_rotate(h0, 7), rigth_rotate(h1, 18));
    std::cout << "c-sum" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << ch[i];
    }*/
   ///return h0;
    while (count < 3)
    {
        __int64** extend_part_message1 = new __int64* [64];
        for (auto i = 0; i < 64; ++i)
        {
            extend_part_message1[i] = new __int64[32]{ 0 };
        }

        if (count == 1)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message1 + (i * 32), 8 * 32);
            }
        }

        if (count == 2)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message2 + (i * 32), 8 * 32);
            }
        }
        std::cout << "extend_part_message" << std::endl;
        for (auto i = 0; i < 64; ++i)
        {
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        /*std::cout << std::endl;
        std::cout << std::endl;
        std::cout << std::endl;*/

        //__int64* str = new __int64[32]{ 0 };

        // __int64 sum_S0_maj = 0;
        // __int64 res_sum_with_round_const = 0;

         //hex_to_dec(round_consts[0]);

        for (auto i = 16; i < 64; ++i)
        {
            s0 = xor_strs_32bit(xor_strs_32bit(rigth_rotate(extend_part_message1[i - 15], 7),
                rigth_rotate(extend_part_message1[i - 15], 18)),
                rigth_shift(extend_part_message1[i - 15], 3));
            s1 = xor_strs_32bit(xor_strs_32bit(rigth_rotate(extend_part_message1[i - 2], 17),
                rigth_rotate(extend_part_message1[i - 2], 19)),
                rigth_shift(extend_part_message1[i - 2], 10));
            //sum_ext1_s0 = 
            //sum_ext2_s1 = ;
            //std::cout << sum_strs_32bit(extend_part_message1[i - 16], s0) << std::endl;
            //std::cout << sum_strs_32bit(extend_part_message1[i - 7], s1) << std::endl;
            sum_ext1_2_s0_s1 = (sum_strs_32bit(extend_part_message1[i - 16], s0) + sum_strs_32bit(extend_part_message1[i - 7], s1)) % 4294967296;
            extend_part_message1[i] = to_binary_32bit(sum_ext1_2_s0_s1);
            /*for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;*/
        }
        std::cout << "extend_part_message_after" << std::endl;
        for (auto i = 0; i < 64; ++i)
        {
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        a = h0;
        b = h1;
        c = h2;
        d = h3;
        e = h4;
        f = h5;
        g = h6;
        h = h7;


        for (auto i = 0; i < 64; ++i)
        {
            S1 = xor_strs_32bit(xor_strs_32bit(rigth_rotate(e, 6), rigth_rotate(e, 11)),
                rigth_rotate(e, 25));
            ch = xor_strs_32bit(and_strs_32bit(e, f), and_strs_32bit(inverse_str_32bit(e), g));
            /* std::cout << "sum_numbers h,S1" << std::endl;
             std::cout << sum_strs_32bit(h, S1) << std::endl;
             std::cout << "sum_numbers ch,xtend_part_message[i]" << std::endl;
             std::cout << sum_strs_32bit(ch, extend_part_message1[i]) << std::endl;
             std::cout << "round_const[i]" << std::endl;
             std::cout << round_consts[i] << std::endl;
             std::cout << "sum" << std::endl;
             std::cout << (sum_strs_32bit(h, S1) + sum_strs_32bit(ch, extend_part_message1[i]) + round_consts[i]) % 4294967296 << std::endl;*/
            temp1 = to_binary_32bit((sum_strs_32bit(h, S1) + sum_strs_32bit(ch, extend_part_message1[i]) + round_consts[i]) % 4294967296);
            /* for (auto i = 0; i < 32; ++i)
             {
                 std::cout << temp1[i];
             }
             std::cout << std::endl;*/
            S0 = xor_strs_32bit(xor_strs_32bit(rigth_rotate(a, 2), rigth_rotate(a, 13)),
                rigth_rotate(a, 22));
            maj = xor_strs_32bit(xor_strs_32bit(and_strs_32bit(a, b), and_strs_32bit(a, c)),
                and_strs_32bit(b, c));
            temp2 = to_binary_32bit(sum_strs_32bit(S0, maj));
            h = g;
            g = f;
            f = e;
            e = to_binary_32bit(sum_strs_32bit(d, temp1));
            d = c;
            c = b;
            b = a;
            a = to_binary_32bit(sum_strs_32bit(temp1, temp2));
        }
        count++;
        delete[] extend_part_message1;
        delete[] s0;
        delete[] s1;
        delete[] S0;
        delete[] S1;
        delete[] ch;
        delete[] temp1;
        delete[] temp2;
        delete[] maj;

        h0 = to_binary_32bit(sum_strs_32bit(h0, a));
        h1 = to_binary_32bit(sum_strs_32bit(h1, b));
        h2 = to_binary_32bit(sum_strs_32bit(h2, c));
        h3 = to_binary_32bit(sum_strs_32bit(h3, d));
        h4 = to_binary_32bit(sum_strs_32bit(h4, e));
        h5 = to_binary_32bit(sum_strs_32bit(h5, f));
        h6 = to_binary_32bit(sum_strs_32bit(h6, g));
        h7 = to_binary_32bit(sum_strs_32bit(h7, h));

    }
    
    __int64* hash = new __int64[256]{ 0 };
    std::cout <<"h0"<< std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h1[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h2[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h3[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h4[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h5[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h6[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h7[i];
    }
    std::cout << std::endl;
    memcpy(hash, h0, 8 * 32);
    memcpy(hash + 32, h1, 8 * 32);
    memcpy(hash + 64, h2, 8 * 32);
    memcpy(hash + 96, h3, 8 * 32);
    memcpy(hash + 128, h4, 8 * 32);
    memcpy(hash + 160, h5, 8 * 32);
    memcpy(hash + 192, h6, 8 * 32);
    memcpy(hash + 224, h7, 8 * 32);
    return hash;

    
    
       

}

__global__ void addKernel(char* binary_pass, int length_block)
{
    
    


}
int main()
{
   
    std::string password = "1234";
    __int64* password_xor_with_ipad = password_xor_with_IPAD("1234", 4);
    __int64* prev_hash_u = new __int64[256]{ 0 };
    __int64* messsage=preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash_u);
    __int64* hash=main_loop_sha256_with_ipad(messsage);
    std::cout << "hash" << std::endl;
    for (auto i = 0; i < 256; ++i)
    {
        std::cout << hash[i];
    }
    /*
    __int64* h0 = new __int64[] { 0, 1, 1, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1 };
    __int64* h1 = new __int64[] { 1, 0, 1, 1, 1, 0, 1, 1, 0, 1, 1, 0, 0, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 1, 0, 1 };
    __int64* res = new __int64[32];
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    res = rigth_shift(h0, 5);
    
    std::cout << "res" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << res[i];
    }*/
    //password_xor_with_IPAD("1234", 4);
    // Add vectors in parallel.
    /*hipError_t cudaStatus = addWithCuda(bin_str_512, 512);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }*/

   /* printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/
   

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(char* binary_pass, int length_block)
{
    /*int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;*/
    //char* dev_hash_output = "1111";
    char* dev_binary_pass = "111";
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_binary_pass, length_block *sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

   /* cudaStatus = hipMalloc((void**)&dev_hash_output, length_block * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }*/

    /*cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }*/

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_binary_pass, binary_pass, length_block * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

   /* cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<16, 16 >>>(dev_binary_pass,length_block);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(binary_pass, dev_binary_pass, length_block * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_binary_pass);
   // hipFree(dev_hash_output);
    //hipFree(dev_b);
    
    return cudaStatus;
}
