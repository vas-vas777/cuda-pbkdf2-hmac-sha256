#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <bitset>
#include <iostream>
//#include <vector>
#include <thrust/host_vector.h>
#include <thrust/copy.h>


hipError_t addWithCuda(char* binary_pass, int length_block);

__global__ void addKernel(char* binary_pass, int length_block)
{
    /*auto h0 = 0x6a09e667;
    auto  h1 = 0xbb67ae85;
    auto  h2 = 0x3c6ef372;
    auto  h3 = 0xa54ff53a;
    auto  h4 = 0x510e527f;
    auto  h5 = 0x9b05688c;
    auto  h6 = 0x1f83d9ab;
    auto  h7 = 0x5be0cd19;
    int round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
                            0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
                            0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
                            0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
                            0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
                            0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
                            0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
                            0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

    int index = threadIdx.x;
    int w[64] = {};*/
    


}
int main()
{
   /* const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };*/
    std::cout << '0' + '1' << std::endl;
    //hipError_t cudaStatus;
    std::string password = "1234";
    std::string binary_str;
    //char* binary_pass = "";
    std::vector<char>hash_output;
    //std::cout << binary_pass << std::endl;
    for (size_t i = 0; i < password.size(); ++i) //перевод в двоичную строку пароля
    {
        binary_str.append(std::bitset<8>(password.c_str()[i]).to_string());
    }
    //binary_pass = &binary_str[0];
    std::vector<char> binary_passwd;
    for (auto i = 0; i < binary_str.size(); ++i)
    {
        binary_passwd.push_back(binary_str[i]);
    }

    //thrust::host_vector<char> binary_passwd = vec_binary_str;

    std::vector<char>pass = binary_passwd;
    pass.push_back('1'); //add 1 
    size_t length_bin_pass = pass.size();
    for (auto i = 0; i < 448 - length_bin_pass; ++i) // add '0' multiple 512 k=448 (mod512)
    {                                               
        pass.push_back('0');
    }
    std::vector<int>length_pass_in_binary;
    int bit;
    size_t number_of_bits_password = binary_passwd.size();
    while (number_of_bits_password != 0)
    {
        bit = number_of_bits_password % 2 ? 1 : 0;
        length_pass_in_binary.push_back(bit);
        number_of_bits_password /= 2;
    }
    for (auto i = 0; i < 64 - length_pass_in_binary.size(); ++i)
    {
        pass.push_back('0');
    }
    for (auto i = 0; i <length_pass_in_binary.size(); ++i)
    {
        pass.push_back((char)(length_pass_in_binary[length_pass_in_binary.size()-1-i])+48);
    }
    
   // std::cout << hash_output.size() << std::endl;
   // char* bin_str_512 = (char*)malloc(hash_output.size());
    for (auto i : pass)
    {
        std::cout << i;
    }
    std::cout << std::endl;
   // thrust::copy(hash_output.begin(), hash_output.end(), &bin_str_512[0]);
    std::string message_schedule[64];
    for (auto i = 0; i < 64; ++i)
    {
        if (i < 16)
        {
            for (auto j = 0; j < 32; ++j)
            {
                message_schedule[i] += pass[j + 32 * i];
            }
        }
        else
        {
            for (auto j = 0; j < 32; ++j)
            {
                message_schedule[i] += '0';
            }
        }
    }
    for (auto i : message_schedule)
    {
        std::cout << i << std::endl;
    }
    std::cout << std::endl;
    char message_schedule_d[64][32];
    for (auto i = 0; i < 64; ++i)
    {
        for (auto j = 0; j < 32; ++j)
        {
            //std::cout << i << " " << j << std::endl;
            message_schedule_d[i][j] = message_schedule[i][j];
        }
    }
    for (auto i = 0; i < 64; ++i)
    {
        for (auto j = 0; j < 32; ++j)
        {
            std::cout << message_schedule_d[i][j];
        }
        std::cout << std::endl;
    }
    // Add vectors in parallel.
    /*hipError_t cudaStatus = addWithCuda(bin_str_512, 512);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }*/

   /* printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/
   

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(char* binary_pass, int length_block)
{
    /*int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;*/
    //char* dev_hash_output = "1111";
    char* dev_binary_pass = "111";
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_binary_pass, length_block *sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

   /* cudaStatus = hipMalloc((void**)&dev_hash_output, length_block * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }*/

    /*cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }*/

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_binary_pass, binary_pass, length_block * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

   /* cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<16, 16 >>>(dev_binary_pass,length_block);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(binary_pass, dev_binary_pass, length_block * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_binary_pass);
   // hipFree(dev_hash_output);
    //hipFree(dev_b);
    
    return cudaStatus;
}
