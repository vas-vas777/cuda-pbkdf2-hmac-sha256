#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <bitset>
#include <iostream>
//#include <vector>
#include <thrust/host_vector.h>
#include <thrust/copy.h>


/*auto h0 = 0x6a09e667;
    auto  h1 = 0xbb67ae85;
    auto  h2 = 0x3c6ef372;
    auto  h3 = 0xa54ff53a;
    auto  h4 = 0x510e527f;
    auto  h5 = 0x9b05688c;
    auto  h6 = 0x1f83d9ab;
    auto  h7 = 0x5be0cd19;
    int round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
                            0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
                            0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
                            0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
                            0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
                            0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
                            0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
                            0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

    int index = threadIdx.x;
    int w[64] = {};*/

hipError_t addWithCuda(char* binary_pass, int length_block);

__host__ __int64* password_xor_with_IPAD(char* password,int length)
{
    __int64* binary_str = new __int64[512]{ 0 };
    __int64 IPAD[] = { 0,0,1,1,0,1,1,0 };
    __int64* ipad = new __int64[512]{ 0 };
    for (auto i = 0; i < 512; ++i)
    {
        if (i < 8 * length)
        {
            binary_str[i] = (0 != (password[i / 8] & 1 << (~i & 7)));
        }
        else
        {
            binary_str[i] = 0;
        }
    }
    for (auto i = 0; i < 512; ++i)
    {
        ipad[i] = IPAD[i % 8];
        
    }
   /* for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << ipad[i];
    }
    std::cout << std::endl;*/
    for (auto i = 0; i < 512; ++i)
    {
        binary_str[i] = binary_str[i] ^ ipad[i];
    }
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    std::cout << sizeof(binary_str) << std::endl;
    return binary_str;
    //std::string binary_str = {};
    //std::string IPAD = {};
    ////std::cout << binary_pass << std::endl;
    //for (auto i = 0; i < 64; ++i) //перевод в двоичную строку пароля
    //{
    //    if (i < password.size())
    //    {
    //        binary_str.append(std::bitset<8>(password.c_str()[i]).to_string());
    //        IPAD.append(std::bitset<8>('0x36').to_string());
    //    }
    //    else
    //    {
    //        binary_str.append(std::bitset<8>(0).to_string());
    //        IPAD.append(std::bitset<8>('0x36').to_string());
    //    }
    //}
    /////std::cout << binary_str << std::endl;
    ////std::cout << std::endl;
    ////std::cout << IPAD << std::endl;
    ////std::cout << binary_str.size() << " " << IPAD.size() << std::endl;
    //binary_str = (std::bitset<512>(binary_str) ^ std::bitset<512>(IPAD)).to_string();
    //return binary_str;
    //std::cout << binary_str << std::endl;
    delete[] binary_str;
    delete[] ipad;
}

__device__ int* password_xor_with_OPAD(char* password, int length)
{
    int* binary_str = new int[512]{ 0 };
    int OPAD [] = { 0,1,0,1,1,1,0,0 };
    int* opad = new int[512]{ 0 };
    for (auto i = 0; i < 512; ++i)
    {
        if (i < 8 * length)
        {
            binary_str[i] = (0 != (password[i / 8] & 1 << (~i & 7)));
        }
        else
        {
            binary_str[i] = 0;
        }
    }
    for (auto i = 0; i < 512; ++i)
    {
        opad[i] = OPAD[i % 8];

    }
    /*for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << opad[i];
    }
    std::cout << std::endl;*/
    for (auto i = 0; i < 512; ++i)
    {
        binary_str[i] = binary_str[i] ^ opad[i];
    }
   /* for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }*/


    return binary_str;
    delete[] binary_str;
    delete[] opad;
    //std::string binary_str = {};
    //std::string IPAD = {};
    ////std::cout << binary_pass << std::endl;
    //for (auto i = 0; i < 64; ++i) //перевод в двоичную строку пароля
    //{
    //    if (i < password.size())
    //    {
    //        binary_str.append(std::bitset<8>(password.c_str()[i]).to_string());
    //        IPAD.append(std::bitset<8>('0x5C').to_string());
    //    }
    //    else
    //    {
    //        binary_str.append(std::bitset<8>(0).to_string());
    //        IPAD.append(std::bitset<8>('0x5C').to_string());
    //    }
    //}
    /////std::cout << binary_str << std::endl;
    ////std::cout << std::endl;
    ////std::cout << IPAD << std::endl;
    ////std::cout << binary_str.size() << " " << IPAD.size() << std::endl;
    //binary_str = (std::bitset<512>(binary_str) ^ std::bitset<512>(IPAD)).to_string();
    //return binary_str;
}

 __host__ void preparation_sha256_with_IPAD(__int64* password_xor_with_ipad, __int64*prev_hash)
{
     __int64* binary_str = new __int64[1024]{ 0 };

    //memcpy(binary_str, password_xor_with_ipad, 512);
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << password_xor_with_ipad[i];
    }
    std::cout << std::endl;

    memcpy(binary_str, password_xor_with_ipad, 8 * 512);
    binary_str[769] = 1;
    binary_str[1014] = 1;
    binary_str[1015] = 1;
    //memmove(binary_str, password_xor_with_ipad, 512);
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
   /* memmove(binary_str+256, prev_hash, 256);
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << binary_str[i];
    }*/

    //std::string binary_str = password_xor_with_ipad;
    //std::string binary_Ui = {};
    //for (auto i = 0; i < Ui.size(); ++i)
    //{
    //    binary_Ui.append(std::bitset<8>(Ui.c_str()[i]).to_string());
    //}
    //binary_str = binary_str + binary_Ui;//512+256 bits=768bits
    //int size_binary_str = binary_str.size();//length of input message (pass xor OPAD)||U(i-1))
    //binary_str.append("1"); //add 1 bit =769 bit
    //for (auto i = 0; i < 191; ++i) // add '0' multiple 512 => 769+191=960 mod 512 = 448 mod 512
    //{
    //    binary_str.append("0");
    //}
    //std::vector<int>length_pass_in_binary = {};//binary vector of size input message  
    //int bit=0;
    ////size_t number_of_bits_password = binary_.size();
    //while (size_binary_str != 0)
    //{
    //    bit = size_binary_str % 2 ? 1 : 0;
    //    length_pass_in_binary.push_back(bit);
    //    size_binary_str /= 2;
    //}
    //for (auto i = 0; i < 64 - length_pass_in_binary.size(); ++i)//add last 64 bits, where 64 - length_pass_in_binary '0'
    //{
    //    binary_str.append("0");
    //}
    //for (auto i = 0; i < length_pass_in_binary.size(); ++i)//add binary vector of value input message
    //{
    //   binary_str.push_back((char)(length_pass_in_binary[length_pass_in_binary.size() - 1 - i]) + 48);
    //}

    //// std::cout << hash_output.size() << std::endl;
    //// char* bin_str_512 = (char*)malloc(hash_output.size());
    //for (auto i : binary_str)//output 1024 binary message
    //{
    //    std::cout << i;
    //}
    //std::cout << std::endl;
    //char **w = new char*[64];
    //for (auto i = 0; i < 64; ++i)
    //    w[i] = new char[32];
    //
    //// thrust::copy(hash_output.begin(), hash_output.end(), &bin_str_512[0]);
    ////std::string message_schedule[64];
    ////for (auto i = 0; i < 64; ++i)
    ////{
    ////    if (i < 16)
    ////    {
    ////        for (auto j = 0; j < 32; ++j)
    ////        {
    ////            message_schedule[i] += pass[j + 32 * i];
    ////        }
    ////    }
    ////    else
    ////    {
    ////        for (auto j = 0; j < 32; ++j)
    ////        {
    ////            message_schedule[i] += '0';
    ////        }
    ////    }
    ////}
    ////for (auto i : message_schedule)
    ////{
    ////    std::cout << i << std::endl;
    ////}
    ////std::cout << std::endl;
    ////char message_schedule_d[64][32];
    //int count=0;
    //for (auto i = 0; i < 64; ++i)
    //{
    //    for (auto j = 0; j < 32; ++j)
    //    {
    //        //std::cout << i << " " << j << std::endl;
    //        w[i][j] = binary_str[count];
    //        count++;
    //    }
    //    count++;
    //}
    //for (auto i = 0; i < 64; ++i)
    //{
    //    for (auto j = 0; j < 32; ++j)
    //    {
    //        std::cout << w[i][j];
    //    }
    //    std::cout << std::endl;
    //}
    //return w;
}

 __host__ void preparation_sha256_with_OPAD(__int64* password_xor_with_opad, __int64* hash)
 {
     __int64* binary_str = new __int64[1024]{ 0 };

     //memcpy(binary_str, password_xor_with_ipad, 512);
     std::cout << std::endl;
     for (auto i = 0; i < 512; ++i)
     {
         std::cout << password_xor_with_opad[i];
     }
     std::cout << std::endl;

     memcpy(binary_str, password_xor_with_opad, 8 * 512);
     binary_str[768] = 1;
     binary_str[1014] = 1;
     binary_str[1015] = 1;
     memcpy(binary_str + 512, hash, 8 * 256);
     //memmove(binary_str, password_xor_with_ipad, 512);
     for (auto i = 0; i < 1024; ++i)
     {
         std::cout << binary_str[i];
     }
     std::cout << std::endl;
 }

__global__ void addKernel(char* binary_pass, int length_block)
{
    
    


}
int main()
{
   
    std::string password = "1234";
    __int64* password_xor_with_ipad = password_xor_with_IPAD("1234", 4);
    __int64* prev_hash_u = new __int64[256]{ 0 };
    preparation_sha256_with_OPAD(password_xor_with_ipad, prev_hash_u);
    //password_xor_with_IPAD("1234", 4);
    // Add vectors in parallel.
    /*hipError_t cudaStatus = addWithCuda(bin_str_512, 512);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }*/

   /* printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/
   

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(char* binary_pass, int length_block)
{
    /*int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;*/
    //char* dev_hash_output = "1111";
    char* dev_binary_pass = "111";
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_binary_pass, length_block * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    /* cudaStatus = hipMalloc((void**)&dev_hash_output, length_block * sizeof(char));
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMalloc failed!");
         goto Error;
     }*/

     /*cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMalloc failed!");
         goto Error;
     }*/

     // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_binary_pass, binary_pass, length_block * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    /* cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMemcpy failed!");
         goto Error;
     }*/

     // Launch a kernel on the GPU with one thread for each element.
    addKernel << <16, 16 >> > (dev_binary_pass, length_block);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(binary_pass, dev_binary_pass, length_block * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_binary_pass);
    // hipFree(dev_hash_output);
     //hipFree(dev_b);

    return cudaStatus;
}
