#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <random>
#include <string>
#include <bitset>
#include <iostream>
#include <fstream>
#include <chrono>


constexpr auto count_passwords = 100;
constexpr auto count_iterations = 10;
constexpr auto length_password = 64;
constexpr auto lenght_hash = 256;
//constexpr auto lenght_hash = 1;



__constant__ const uint32_t round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };






__device__ uint32_t* to_binary_32bit(uint32_t number, uint32_t* binary_str)
{
    int count = 31;
    while (number != 0)
    {

        binary_str[count] = number % 2;

        number /= 2;
        count--;

    }
    
  return binary_str;
   
}

__device__ uint32_t str_to_32bitnumber(uint32_t* str)
{
    uint32_t number1 = 0;
    for (auto i = 0; i < 32; ++i)
    {
        uint32_t number2 = 1;
        for (auto j = 0; j < (31 - i); ++j)
        {
            number2 *= 2;
        }
        number1 += str[i] * number2;
    }

    return number1;
}



__device__ uint32_t sum_strs_32bit(uint32_t *str1, uint32_t *str2)
{
    uint32_t number1[1]{ 0 };
    uint32_t number2[1]{ 0 };
   
    uint32_t res_number = 0;
    for (auto i = 0; i < 32; ++i)
    {
        atomicAdd(&number1[0], str1[i] * __powf(2, (31 - i)));
        atomicAdd(&number2[0], str2[i] * __powf(2, (31 - i)));
    }
    res_number = ((number1[0]) + (number2[0])) % 4294967296;
    return res_number;
   
}


__device__ void xor_strs(uint32_t* str1, uint32_t* str2, unsigned int length, uint32_t* result_xor, uint32_t* str3)
{
   
    for (auto i = 0; i <length; ++i)   
    {
        result_xor[i] = str1[i] ^ str2[i];
        result_xor[i] = result_xor[i] ^ str3[i];
    }
    
}




__device__ void password_xor_with_IPAD(uint32_t* password,size_t length, uint32_t* output_str)
{
    uint32_t binary_str[512]{ 0 };
    memcpy(binary_str, password, sizeof(uint32_t) * length_password);
    
    uint32_t IPAD[] = { 0,0,1,1,0,1,1,0 };

  
    
    for (int i = 0; i < 512; ++i)
    {
        output_str[i] = binary_str[i] ^ IPAD[i % 8];
      //  __syncthreads();
    }
  //  __syncthreads();
    
}

__device__ void password_xor_with_OPAD(uint32_t* password, size_t length, uint32_t *output_str)
{
    uint32_t binary_str[512]{ 0 };
    uint32_t OPAD[] = { 0,1,0,1,1,1,0,0 };
    memcpy(binary_str, password, sizeof(uint32_t) * length_password);
   
   // __syncthreads();
    for (int k = 0; k < 512; ++k)
    {
        output_str[k] = binary_str[k] ^ OPAD[k % 8];
       // __syncthreads();
    }
   // __syncthreads();
}

 __device__ void preparation_sha256_with_IPAD(uint32_t* password_xor_with_ipad, uint32_t*prev_hash, uint32_t *output_str)
{
     uint32_t message[1024]{ 0 };
     for (int k = 0; k < 512; ++k)
     {
         
         message[k] = password_xor_with_ipad[k];
        // __syncthreads();
     }

     for (int k = 0; k < 256; ++k)
     {
         message[k + 512] = prev_hash[k];
       //  __syncthreads();
     }
     // __syncthreads();
     message[768] = 1;
     message[1014] = 1;
     message[1015] = 1;
     for (int k = 0; k < 1024; ++k)
     {
         output_str[k] = message[k];
       //  __syncthreads();
     }
}

__device__ void preparation_sha256_with_OPAD(uint32_t* password_xor_with_opad, uint32_t* prev_hash, uint32_t* output_str)
{
    uint32_t message[1024]{ 0 };
    for (int k = 0; k < 512; ++k)
    {
        
        message[k] = password_xor_with_opad[k];
       // __syncthreads();
    }

    for (int k = 0; k < 256; ++k)
    {
       
        message[k + 512] = prev_hash[k];
       // __syncthreads();
    }

    message[768] = 1;
    message[1014] = 1;
    message[1015] = 1;
    for (int j = 0; j < 1024; ++j)
    {

        output_str[j] = message[j];
       // __syncthreads();
    }
    
   
}

__device__ void main_loop_sha256(uint32_t* message, uint32_t* output_hash)
{

    uint32_t h0[32]{ 0, 1, 1, 0,  1, 0, 1, 0,  0, 0, 0, 0,  1, 0, 0, 1,  1, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 1 };
    uint32_t h1[32]{ 1, 0, 1, 1,  1, 0, 1, 1,  0, 1, 1, 0,  0, 1, 1, 1,  1, 0, 1, 0,  1, 1, 1, 0,  1, 0, 0, 0,  0, 1, 0, 1 };
    uint32_t h2[32]{ 0, 0, 1, 1,  1, 1, 0, 0,  0, 1, 1, 0,  1, 1, 1, 0,  1, 1, 1, 1,  0, 0, 1, 1,  0, 1, 1, 1,  0, 0, 1, 0 };
    uint32_t h3[32]{ 1, 0, 1, 0,  0, 1, 0, 1,  0, 1, 0, 0,  1, 1, 1, 1,  1, 1, 1, 1,  0, 1, 0, 1,  0, 0, 1, 1,  1, 0, 1, 0 };
    uint32_t h4[32]{ 0, 1, 0, 1,  0, 0, 0, 1,  0, 0, 0, 0,  1, 1, 1, 0,  0, 1, 0, 1,  0, 0, 1, 0,  0, 1, 1, 1,  1, 1, 1, 1 };
    uint32_t h5[32]{ 1, 0, 0, 1,  1, 0, 1, 1,  0, 0, 0, 0,  0, 1, 0, 1,  0, 1, 1, 0,  1, 0, 0, 0,  1, 0, 0, 0,  1, 1, 0, 0 };
    uint32_t h6[32]{ 0, 0, 0, 1,  1, 1, 1, 1,  1, 0, 0, 0,  0, 0, 1, 1,  1, 1, 0, 1,  1, 0, 0, 1,  1, 0, 1, 0,  1, 0, 1, 1 };
    uint32_t h7[32]{ 0, 1, 0, 1,  1, 0, 1, 1,  1, 1, 1, 0,  0, 0, 0, 0,  1, 1, 0, 0,  1, 1, 0, 1,  0, 0, 0, 1,  1, 0, 0, 1 };
       
    uint32_t part_message1[512]{ 0 };
    uint32_t part_message2[512]{ 0 };
    memcpy(part_message1, message, 4 * 512);
    memcpy(part_message2, message + 512, 4 * 512);

    int count = 1;//счётчик для 2ух итераций



    while (count < 3)
    {


     
        uint32_t temp1[32]{ 0 };
        uint32_t temp2[32]{ 0 };

        uint32_t a[32]{ 0 };
        uint32_t b[32]{ 0 };
        uint32_t c[32]{ 0 };
        uint32_t d[32]{ 0 };
        uint32_t e[32]{ 0 };
        uint32_t f[32]{ 0 };
        uint32_t g[32]{ 0 };
        uint32_t h[32]{ 0 };
        uint32_t extend_part_message1[64][32]{ 0 };
        



        if (count == 1)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message1 + (i * 32), 4 * 32);
            }
        }

        if (count == 2)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message2 + (i * 32), 4 * 32);
            }
        }

        for (auto i = 16; i < 64; ++i)
        {
            for (int j = 0; j < 32; j++)
            {
                extend_part_message1[i][j] = extend_part_message1[i - 16][j] + extend_part_message1[i - 7][j] +
                    ((extend_part_message1[i - 15][(j + 25) % 32] + extend_part_message1[i - 15][(j + 14) % 32] + (j < 3 ? 0 : extend_part_message1[i - 15][(j + 29) % 32])) % 2) +
                    ((extend_part_message1[i - 2][(j + 15) % 32] + extend_part_message1[i - 2][(j + 13) % 32] + (j < 10 ? 0 : extend_part_message1[i - 2][(j + 22) % 32])) % 2);
            }
            for (int j = 31; j > 0; j--)
            {
                while (extend_part_message1[i][j] >= 2) {
                    extend_part_message1[i][j] -= 2;
                    extend_part_message1[i][j - 1]++;
                }
            }
            extend_part_message1[i][0] = extend_part_message1[i][0] % 2;
           
          

        }
       
       
        memcpy(a, h0, 4 * 32);
        memcpy(b, h1, 4 * 32);
        memcpy(c, h2, 4 * 32);
        memcpy(d, h3, 4 * 32);
        memcpy(e, h4, 4 * 32);
        memcpy(f, h5, 4 * 32);
        memcpy(g, h6, 4 * 32);
        memcpy(h, h7, 4 * 32);

        for (auto i = 0; i < 64; ++i)
        {
                                        
            uint32_t round_cnst[32]{ 0 };
            to_binary_32bit(round_consts[i], round_cnst);
           
            for (int j = 0; j < 32; j++)
            {
                temp2[j] = ((a[(j + 30) % 32] + a[(j + 19) % 32] + a[(j + 10) % 32]) % 2) +
                    (((b[j] == 1 && a[j] == 1 ? 1 : 0) + (c[j] == 1 && a[j] == 1 ? 1 : 0) + (c[j] == 1 && b[j] == 1 ? 1 : 0)) % 2);

                temp1[j] = h[j] +
                    ((e[(j + 26) % 32] + e[(j + 21) % 32] + e[(j + 7) % 32]) % 2) +
                    (((f[j] == 1 && e[j] == 1 ? 1 : 0) + (e[j] == 0 && g[j] == 1 ? 1 : 0)) % 2) +
                    extend_part_message1[i][j] +
                    round_cnst[j];
            }

            for (int j = 31; j > 0; j--)
            {
                while (temp2[j] >= 2) {
                    temp2[j] -= 2;
                    temp2[j - 1]++;
                }

                while (temp1[j] >= 2) {
                    temp1[j] -= 2;
                    temp1[j - 1]++;
                }
            }
            temp2[0] = temp2[0] % 2;
            temp1[0] = temp1[0] % 2;


       
            uint32_t to_binary_32bit2[32]{ 0 };
            uint32_t to_binary_32bit3[32]{ 0 };
           

           
            

            memcpy(h, g, 4 * 32);
           

            memcpy(g, f, 4 * 32);
            


            memcpy(f, e, 4 * 32);
          

            to_binary_32bit(sum_strs_32bit(d, temp1), to_binary_32bit2);
            memcpy(e, to_binary_32bit2, 4 * 32);
           

            memcpy(d, c, 4 * 32);
           
            memcpy(c, b, 4 * 32);
            
            memcpy(b, a, 4 * 32);
           

            to_binary_32bit(sum_strs_32bit(temp1, temp2),to_binary_32bit3);
            memcpy(a, to_binary_32bit3, 4 * 32);        

        }
         
        count++;

        uint32_t to_binary_32bith0[32]{ 0 };
        uint32_t to_binary_32bith1[32]{ 0 };
        uint32_t to_binary_32bith2[32]{ 0 };
        uint32_t to_binary_32bith3[32]{ 0 };
        uint32_t to_binary_32bith4[32]{ 0 };
        uint32_t to_binary_32bith5[32]{ 0 };
        uint32_t to_binary_32bith6[32]{ 0 };
        uint32_t to_binary_32bith7[32]{ 0 };

        to_binary_32bit(sum_strs_32bit(h0, a), to_binary_32bith0);
        memcpy(h0, to_binary_32bith0, 4 * 32);
        to_binary_32bit(sum_strs_32bit(h1, b), to_binary_32bith1);
        memcpy(h1, to_binary_32bith1, 4 * 32);
        to_binary_32bit(sum_strs_32bit(h2, c), to_binary_32bith2);
        memcpy(h2, to_binary_32bith2, 4 * 32);
        to_binary_32bit(sum_strs_32bit(h3, d), to_binary_32bith3);
        memcpy(h3, to_binary_32bith3, 4 * 32);
        to_binary_32bit(sum_strs_32bit(h4, e), to_binary_32bith4);
        memcpy(h4, to_binary_32bith4, 4 * 32);
        to_binary_32bit(sum_strs_32bit(h5, f), to_binary_32bith5);
        memcpy(h5, to_binary_32bith5, 4 * 32);
        to_binary_32bit(sum_strs_32bit(h6, g), to_binary_32bith6);
        memcpy(h6, to_binary_32bith6, 4 * 32);
        to_binary_32bit(sum_strs_32bit(h7, h), to_binary_32bith7);
        memcpy(h7, to_binary_32bith7, 4 * 32);
       
    }

    uint32_t hash[256]{ 0 };
    memcpy(hash, h0, 4 * 32);
    memcpy(hash + 32, h1, 4 * 32);
    memcpy(hash + 64, h2, 4 * 32);
    memcpy(hash + 96, h3, 4 * 32);
    memcpy(hash + 128, h4, 4 * 32);
    memcpy(hash + 160, h5, 4 * 32);
    memcpy(hash + 192, h6, 4 * 32);
    memcpy(hash + 224, h7, 4 * 32);
    memcpy(output_hash, hash, 4 * 256);
   

}



//__device__ void found_pass(uint32_t *password, int k)
//{
//    for (size_t i = 0; i < 64; i++)
//    {
//        password[i] = password[i] + k;
//        printf("%u", password[i]);
//    }
//}




__global__ void pbkdf2_hmac_sha256(unsigned int c, 
    uint32_t* password, size_t length, uint32_t* salt, uint32_t* search_hash_pbkdf2, uint32_t* pbkdf2_hashes)
{
   
     uint32_t message[1024]{ 0 };
     uint32_t prev_hash_hmac[256]{ 0 };
     uint32_t zero_str[256]{ 0 };
     uint32_t part_pbkdf2_hash[lenght_hash]{ 0 };
     uint32_t password_xor_ipad[512]{ 0 };
     uint32_t password_xor_opad[512]{ 0 };
     uint32_t current_password[64]{ 0 };
    // bool password_found = false;
    

     for (int k = blockIdx.x * blockDim.x + threadIdx.x;
         k < count_passwords;
         k += blockDim.x * gridDim.x)
     {
         
         memcpy(current_password, password + k * length_password, sizeof(uint32_t) * length_password);
         

         password_xor_with_IPAD(current_password, length, password_xor_ipad);
       //  __syncthreads();
         password_xor_with_OPAD(current_password, length, password_xor_opad);
       //  __syncthreads();
         memcpy(prev_hash_hmac, salt, sizeof(uint32_t) * 256);
       //  __syncthreads();
         for (auto j = 0; j < c; ++j)
         {
             

             preparation_sha256_with_IPAD(password_xor_ipad, prev_hash_hmac, message);
           //  __syncthreads();
             main_loop_sha256(message, prev_hash_hmac);
           //  __syncthreads();
             preparation_sha256_with_OPAD(password_xor_opad, prev_hash_hmac, message);
           //  __syncthreads();
             main_loop_sha256(message, prev_hash_hmac);
           //  __syncthreads();
             xor_strs(prev_hash_hmac, part_pbkdf2_hash, 256, part_pbkdf2_hash, zero_str);
          //   __syncthreads();
             //  __threadfence();
               //  __threadfence();
         }
        // __syncthreads();
         memcpy(pbkdf2_hashes + k * 256, part_pbkdf2_hash, sizeof(uint32_t) * 256);
       //  __syncthreads();

        
     }
        

}


__global__ void search_current_hash_in_hashes(uint32_t * search_hash_pbkdf2, uint32_t* pbkdf2_hashes, uint32_t* position_found)
{
  //  uint32_t temp_hash_pbkdf2[lenght_hash]{ 0 };
    //auto flag = false;

    for (int k = blockIdx.x * blockDim.x + threadIdx.x;
        k < count_passwords*lenght_hash;
        k += blockDim.x * gridDim.x)
    {
      //  memcpy(temp_hash_pbkdf2, pbkdf2_hashes + blockIdx.x * lenght_hash, sizeof(uint32_t) * lenght_hash);
        
        if (pbkdf2_hashes[k] != search_hash_pbkdf2[threadIdx.x])
        {
           position_found[blockIdx.x] = count_passwords;
          // flag = true;
        }
      //  printf("\npos=%d", position_found[43]);
        if (position_found[blockIdx.x] == 0)
        {
            position_found[0] = blockIdx.x;
        }
      
    }
}



//uint32_t* random_salt(size_t Nbits)
//{
//    std::random_device rd;  //Will be used to obtain a seed for the random number engine
//    std::mt19937 gen(rd());
//    std::uniform_int_distribution<> int1(0, 1);
//    uint32_t* str = new uint32_t[Nbits];
//    //str.reserve(Nbits);
//    for (size_t i = 0; i < Nbits; i++)
//    {
//        str[i] = int1(gen) ? 1 : 0;
//    }
//    return str;
//};



int main()
{


    unsigned int len_hash_pbkdf2 = 2;
    uint32_t* salt = new uint32_t[256]{ 0 };

    std::ifstream file("8digits.txt");
    std::string pass{ 0 };
    std::string list_of_passwords;
    uint32_t binary_passwords[length_password * count_passwords]{ 0 };
    int count = 0;

    std::string bin_pass;
    std::cout << "passwords" << std::endl;
    if (file.is_open())
    {
        while (!file.eof() && count < length_password * count_passwords)
        {
            file >> pass;
            list_of_passwords.append(pass);
            std::cout << "count=" << count / 64 << " " << pass << std::endl;
            for (std::size_t i = 0; i < pass.size(); ++i)
            {
                // std::cout << std::bitset<8>(pass.c_str()[i]).to_string();
                bin_pass.append(std::bitset<8>(pass.c_str()[i]).to_string());
            }
            // std::cout << std::endl;
            for (std::size_t j = 0; j < bin_pass.size(); ++j)
            {
                binary_passwords[count] = uint32_t(bin_pass[j]) - 48;
                count++;
            }
            bin_pass.erase();

        }

    }
    std::cout << std::endl;
    //auto t2 = std::chrono::high_resolution_clock::now();
    for (auto i = 0; i < length_password * count_passwords; ++i)
    {

        if (i % 64 == 0)
        {
            std::cout << std::endl;
            std::cout << "count i=" << i / 64 << " ";
            std::cout << binary_passwords[i];

        }
        else
        {
            std::cout << binary_passwords[i];
        }


        // std::cout << std::endl;
    }
    std::cout << std::endl;
    //std::cout << std::endl;
  //  return 0;




    uint32_t pbkdf2_hash[count_passwords * lenght_hash]{ 0 };
    uint32_t search_hash_pbkdf2[lenght_hash]{ 0,0,1,1,1,1,0,0,0,1,1,1,0,1,0,1,0,0,1,1,0,1,1,0,1,1,0,0,1,
        1,0,0,1,0,1,1,1,0,1,0,0,1,1,0,0,1,0,1,1,0,0,0,0,1,0,1,1,0,0,1,1,0,1,0,1,1,1,1,0,1,0,0,1,0,1,0,0,0,1,0,0,1,1,1,1,1,1,1,
        0,1,1,0,0,1,0,0,0,0,1,1,1,0,0,1,1,1,1,0,0,0,0,1,0,0,0,1,0,1,1,1,0,1,1,1,0,0,0,1,1,0,0,1,1,1,0,1,1,0,1,0,0,1,1,0,1,0,1,
        0,1,0,1,1,1,1,1,0,1,0,1,1,0,1,1,1,1,0,1,0,0,0,0,1,0,0,1,1,0,1,1,0,0,0,0,0,0,1,1,1,1,1,0,0,1,0,0,1,1,0,0,1,0,1,0,0,0,0,
        0,0,0,1,0,0,1,0,1,0,0,1,1,0,0,1,0,0,1,1,1,0,0,1,1,1,1,1,0,1,0,1,1,0,1,1,1,0,1,0,1,1,1,1,0,1,0,1,0,0, };
    uint32_t found_password[length_password]{ 0 };
    uint32_t hash_found[lenght_hash]{ 0 };
    uint32_t* position_found[1]{ 0 };
    uint32_t* dev_password_xor_with_ipad = nullptr;
    uint32_t* dev_password_xor_with_opad = nullptr;
    uint32_t* dev_pbkdf2_hash = nullptr;
    uint32_t* dev_search_hash_pbkdf2 = nullptr;
    uint32_t* dev_salt = nullptr;
    uint32_t* dev_binary_password = nullptr;
    uint32_t* dev_hash_found = nullptr;
    uint32_t* dev_position_found = nullptr;

    /*hipDeviceProp_t deviceProp;*/
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    /*hipGetDeviceProperties(&deviceProp, 0);
    std::cout << deviceProp.maxThreadsPerBlock << std::endl;
    std::cout << std::hex<<deviceProp.maxThreadsDim << std::endl;*/
    

    cudaStatus = hipMalloc((void**)&dev_binary_password, length_password * count_passwords * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }
    cudaStatus = hipMemcpy(dev_binary_password, binary_passwords, length_password * count_passwords * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

        cudaStatus = hipMalloc((void**)&dev_pbkdf2_hash, count_passwords * lenght_hash * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_pbkdf2_hash, pbkdf2_hash, count_passwords * lenght_hash * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

   
    cudaStatus = hipMalloc((void**)&dev_salt, 256 * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_salt, salt, 256 * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    pbkdf2_hmac_sha256 << <count_passwords, 1 >> > (count_iterations,
        dev_binary_password, 8, dev_salt, dev_search_hash_pbkdf2, dev_pbkdf2_hash);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
 
    cudaStatus = hipMalloc((void**)&dev_position_found, sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_position_found, position_found, sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_search_hash_pbkdf2, lenght_hash * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_search_hash_pbkdf2, search_hash_pbkdf2, lenght_hash * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    
    hipEventRecord(start);

   search_current_hash_in_hashes << <count_passwords, lenght_hash >> > (dev_search_hash_pbkdf2, dev_pbkdf2_hash, dev_position_found);

    hipEventRecord(stop);


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    hipEventRecord(stop);
    cudaStatus = hipMemcpy(position_found, dev_position_found, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "milliseconds=" << milliseconds << std::endl;

   


   
  //  std::cout << "position=" << int(position_found[0]) << std::endl;
    if ((int)position_found[0] == count_passwords)
    {
        std::cout << "password not found" << std::endl;
    }
    else
    {
        std::cout << "position=" << int(position_found[0]) << std::endl;
        std::cout << "password" << std::endl;
        for (auto i = int(position_found[0]) * 8; i < int(position_found[0]) * 8 + 8; ++i)
        {
            std::cout << list_of_passwords[i];
        }
        std::cout << std::endl;
    }
    /*cudaStatus = hipMemcpy(pbkdf2_hash, dev_pbkdf2_hash, count_passwords * 256 * sizeof(uint32_t), hipMemcpyDeviceToHost);
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMemcpy failed!");
         goto Error;
     }

   
     auto found_pass = false;
     uint32_t temp_hash[256]{ 0 };
    auto t1 = std::chrono::high_resolution_clock::now();
    for (auto i = 0; i < count_passwords; ++i)
    {
        memcpy(temp_hash, pbkdf2_hash + i * 256, sizeof(uint32_t) * 256);
        if (std::equal(std::begin(temp_hash), std::end(temp_hash), std::begin(search_hash_pbkdf2)))
        {
            std::cout << "number-password=" << i << std::endl;
            memcpy(found_password, binary_passwords + 64 * i, sizeof(uint32_t) * 64);
            std::copy(bin_pass.begin(), bin_pass.end(), found_password);
            for (auto j = 0; j < 64; ++j)
            {
                std::cout << found_password[j];
            }
            std::cout << std::endl;
            std::cout << "searched_hash" << std::endl;
            for (auto j = 0; j < 256; ++j)
            {
                std::cout << temp_hash[j];
            }
            found_pass = true;
            break;
        }
        

    }
    auto t2 = std::chrono::high_resolution_clock::now();
    std::cout << "CPU time: "
        << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count()
        << "ms" << std::endl;


    if (found_pass == false)
    {
        std::cout << "password not found" << std::endl;
    }*/
    //std::cout << std::endl;
    //std::cout << "Hashes" << std::endl;
    //for (auto i = 0; i < count_passwords * 256; ++i)
    //{

    //    if ((i % 256) == 0)
    //    {
    //        std::cout << std::endl;
    //        std::cout << "count i=" << i / (256) << " ";
    //        std::cout << pbkdf2_hash[i];
    //        // std::cout << std::endl;
    //    }
    //    else
    //    {
    //        std::cout << pbkdf2_hash[i];
    //    }


    ////    // std::cout << std::endl;
    //}
    //std::cout << std::endl;


    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    hipFree(dev_password_xor_with_ipad);
    hipFree(dev_password_xor_with_opad);
    hipFree(dev_pbkdf2_hash);
    hipFree(dev_salt);
    hipFree(dev_binary_password);

    return 0;
Error:
    hipFree(dev_password_xor_with_ipad);
    hipFree(dev_password_xor_with_opad);
    hipFree(dev_pbkdf2_hash);
    hipFree(dev_salt);
    hipFree(dev_binary_password);


}
