#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <string>
#include <iostream>
#include <bitset>
#include<thrust/copy.h>



//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}

int main()
{
    /*const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };*/
    thrust::host_vector<char> hash_output;
    //hipError_t cudaStatus;
    std::string password = "1234";
    std::string binary_str;
    for (size_t i = 0; i < password.size(); ++i)
    {
        binary_str.append(std::bitset<8>(password.c_str()[i]).to_string());
    }

    thrust::host_vector<char> vec_binary_str;
    for (auto i = 0; i < binary_str.size(); ++i)
    {
        vec_binary_str.push_back(binary_str[i]);
    }

    thrust::device_vector<char> binary_passwd = vec_binary_str;

    thrust::device_vector<char>pass = binary_passwd;
    pass.push_back('1'); //add 1 
    for (auto i = 0; i < 447 - pass.size(); ++i) // add '0' multiple 512 
    {
        pass.push_back('0');
    }
    thrust::device_vector<char>length_pass_in_binary;
    char bit;
    size_t number_of_bits_password = binary_passwd.size();
    while (number_of_bits_password != 0)
    {
        bit = number_of_bits_password % 2 ? 1 : 0;
        length_pass_in_binary.push_back(bit);
        number_of_bits_password /= 2;
    }
    for (auto i = 0; i < 63 - length_pass_in_binary.size(); ++i)
    {
        pass.push_back('0');
    }
    for (auto i = length_pass_in_binary.size() - 1; i > 0; --i)
    {
        pass.push_back(length_pass_in_binary[i]);
    }
    hash_output = pass;

    //addWithCuda(binary_passwd, binary_passwd.size(), hash_output);
       /* if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }*/
        // }
    for (auto i : hash_output)
    {
        std::cout << i;

    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    // Add vectors in parallel.
   /* hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
   /* cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
