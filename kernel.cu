#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include <string>
#include <bitset>
#include <iostream>
#include <math.h>
//#include <openssl/hmac.h>


hipError_t addWithCuda(char* binary_pass, int length_block);

__constant__ const uint32_t round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

//__device__ uint32_t pow_uint32_t(uint32_t a, uint32_t x)
//{
//    uint32_t res = 1;
//    for (auto i = 0; i < x; ++i)
//    {
//        res = a * res;
//    }
//    return res;
//}

__device__ uint32_t* to_binary_32bit(uint32_t number)
{
    uint32_t *binary_str=new uint32_t [32]{ 0 };

    //int count = threadIdx.x + blockIdx.x;
    int count = 31;
    //std::cout << number << std::endl;
    while (number!=0)
    {
       
        binary_str[count] = number % 2;
        
        number /= 2;
        count--;
       
    }
   // __syncthreads();
    
    return binary_str;
    //delete[] binary_str;
}

__device__ uint32_t str_to_32bitnumber(uint32_t* str)
{
    uint32_t number1 = 0;
    //
   // int i = threadIdx.x ;
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (auto i = 0; i < 32; ++i)
    {
        uint32_t number2 = 1;
        for (auto j = 0; j < (31 - i); ++j)
        {
            number2 *= 2;
        }
        number1 += str[i] * number2;
    }

    return number1;
}

__device__ uint32_t* and_strs_32bit(uint32_t* str1, uint32_t* str2)
{
    uint32_t* result_str = new uint32_t[32]{ 0 };
   // int i = blockIdx.x * blockDim.x + threadIdx.x;
   // int i = threadIdx.x;
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = str1[i] & str2[i];
    }
  //  __syncthreads();
    return result_str;
    //delete[] result_str;
}

__device__ uint32_t* inverse_str_32bit(uint32_t* str1)
{
    uint32_t* result_str = new uint32_t[32]{ 0 };
   
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = !str1[i];
        //printf("%u", result_str[i]);
    }
        //__syncthreads();
    return result_str;
}

__device__ uint32_t* inverse_str_256bit(uint32_t* str1)
{
    uint32_t* result_str = new uint32_t[256]{ 0 };

   
   // int i = threadIdx.x ;
    //if (i < 256)
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //int i = threadIdx.x;
    for (auto i = 0; i < 256; ++i)
    {
        result_str[i] = !str1[i];
        
    }
   // __syncthreads();
    return result_str;
    
}

__device__ uint32_t sum_strs_32bit(uint32_t str1[], uint32_t str2[])
{
    uint32_t number1 = 0;
    uint32_t number2 = 0;
    uint32_t res_number = 0;
    // int i = threadIdx.x ;
    for (int i = 0; i < 32; ++i)
        // if(i<32)
         //int i = blockIdx.x * blockDim.x + threadIdx.x;
    {
        uint32_t number3 = 1;
        for (auto j = 0; j < (31 - i); ++j)
        {
            number3 *= 2;
        }
        number1 += str1[i] * number3;
    }
    for (int i = 0; i < 32; ++i)
    {
        uint32_t number3 = 1;
        for (auto j = 0; j < (31 - i); ++j)
        {
            number3 *= 2;
        }
        number2 += str2[i] * number3;
    }
    res_number = (number1 + number2) % 4294967296;
    __syncthreads();
    return res_number;
    // delete[]result_sum;
     //delete[]result_sum;
}

__device__ uint32_t* xor_strs(uint32_t* str1, uint32_t* str2, unsigned int length)
{
    uint32_t* result_xor = new uint32_t[length]{ 0 };
   // int i = threadIdx.x ;
    for (auto i = 0; i < length; ++i)
    //int i = threadIdx.x;
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    {
        result_xor[i] = str1[i] ^ str2[i];
    }
   // __syncthreads();
    return result_xor;
    //delete[]result_xor;
}

__device__ uint32_t rigth_rotate(uint32_t* str, unsigned int num)
{
    uint32_t number = 0;
    uint32_t* rotated_str = new uint32_t[32]{ 0 };
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //int i = threadIdx.x;
    //rotated_str[i] = str[i];
    memcpy(rotated_str, str, 4 * 32);
    number = str_to_32bitnumber(rotated_str);
   // __syncthreads();
    return (number >> num | number << (32 - num));
    
}

__device__ uint32_t rigth_shift(uint32_t* str, unsigned int num)
{
    
    uint32_t* shifted_str = new uint32_t[32]{ 0 };
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //shifted_str[i] = str[i];
   memcpy(shifted_str, str, 4 * 32);
   // int i = threadIdx.x;
  //  shifted_str[i] = str[i];
    uint32_t number = str_to_32bitnumber(shifted_str);
   // delete[] shifted_str;
   // __syncthreads();
    return number >> num;
       
}



__device__ void password_xor_with_IPAD(char* password,unsigned int length, uint32_t* output_str)
{
    uint32_t *binary_str= new uint32_t[512]{ 0 };
    //uint32_t* output_str = new uint32_t[512]{ 0 };
    uint32_t IPAD[] = { 0,0,1,1,0,1,1,0 };
    //uint32_t *ipad = new uint32_t[512]{ 0 };
    //for (auto i = 0; i < 512; ++i)
    int i = threadIdx.x;
    //while (i<512)
    //{
    output_str[i] = binary_str[i] ^ IPAD[i % 8];
   // __syncthreads();
    //}
    //IPAD[i%8];
    //return output_str;
  
    
}

__device__ uint32_t* password_xor_with_OPAD(char* password, unsigned int length, uint32_t *output_str)
{
    uint32_t *binary_str = new uint32_t[512]{ 0 };
    uint32_t OPAD [] = { 0,1,0,1,1,1,0,0 };
   // uint32_t *opad = new uint32_t[512]{ 0 };
   // for (auto i = 0; i < 512; ++i)
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int i = threadIdx.x;
    output_str[i] = binary_str[i] ^ OPAD[i % 8];
    __syncthreads();
    //return output_str;
    
}

 __device__ void preparation_sha256_with_IPAD(uint32_t* password_xor_with_ipad, uint32_t*prev_hash, uint32_t *output_str)
{
     uint32_t *binary_str = new uint32_t[1024]{ 0 };
     memcpy(binary_str, password_xor_with_ipad, 4 * 512);
     memcpy(binary_str + 512, prev_hash, 4 * 256);
     binary_str[768] = 1;
     binary_str[1014] = 1;
     binary_str[1015] = 1;
     memcpy(output_str, binary_str, 4 * 1024);
     //return output_str;
}

__device__ uint32_t* preparation_sha256_with_OPAD(uint32_t* password_xor_with_opad, uint32_t* prev_hash)
{
    uint32_t *binary_str= new uint32_t[1024]{ 0 };
  

    memcpy(binary_str, password_xor_with_opad, 4 * 512);
    memcpy(binary_str + 512, prev_hash, 4 * 256);
    binary_str[768] = 1;
    binary_str[1014] = 1;
    binary_str[1015] = 1;
   
    return binary_str;
   
}

__device__ void main_loop_sha256(uint32_t* message, uint32_t* output_hash)
{
   
    uint32_t h0[32] { 0, 1, 1, 0,  1, 0, 1, 0,  0, 0, 0, 0,  1, 0, 0, 1,  1, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 1 };
    uint32_t h1[32] { 1, 0, 1, 1,  1, 0, 1, 1,  0, 1, 1, 0,  0, 1, 1, 1,  1, 0, 1, 0,  1, 1, 1, 0,  1, 0, 0, 0,  0, 1, 0, 1 };
    uint32_t h2[32] { 0, 0, 1, 1,  1, 1, 0, 0,  0, 1, 1, 0,  1, 1, 1, 0,  1, 1, 1, 1,  0, 0, 1, 1,  0, 1, 1, 1,  0, 0, 1, 0 };
    uint32_t h3[32] { 1, 0, 1, 0,  0, 1, 0, 1,  0, 1, 0, 0,  1, 1, 1, 1,  1, 1, 1, 1,  0, 1, 0, 1,  0, 0, 1, 1,  1, 0, 1, 0 };
    uint32_t h4[32] { 0, 1, 0, 1,  0, 0, 0, 1,  0, 0, 0, 0,  1, 1, 1, 0,  0, 1, 0, 1,  0, 0, 1, 0,  0, 1, 1, 1,  1, 1, 1, 1 };
    uint32_t h5[32] { 1, 0, 0, 1,  1, 0, 1, 1,  0, 0, 0, 0,  0, 1, 0, 1,  0, 1, 1, 0,  1, 0, 0, 0,  1, 0, 0, 0,  1, 1, 0, 0 };
    uint32_t h6[32] { 0, 0, 0, 1,  1, 1, 1, 1,  1, 0, 0, 0,  0, 0, 1, 1,  1, 1, 0, 1,  1, 0, 0, 1,  1, 0, 1, 0,  1, 0, 1, 1 };
    uint32_t h7[32] { 0, 1, 0, 1,  1, 0, 1, 1,  1, 1, 1, 0,  0, 0, 0, 0,  1, 1, 0, 0,  1, 1, 0, 1,  0, 0, 0, 1,  1, 0, 0, 1 };
    uint32_t part_message1[512]{ 0 };/*{ 0, 1, 1, 0, 1, 0, 0, 0, 0, 1, 1, 0, 0, 1, 0, 1, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 1, 1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 1, 1, 0, 1, 1, 0, 1, 1, 1, 1,
0,1,1,1,0,0,1,0, 0,1,1,0,1,1,0,0, 0,1,1,0,0,1,0,0, 1,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,1,0,1,1,0,0,0 };*/
    uint32_t part_message2[512]{ 0 };
    memcpy(part_message1, message, 4 * 512);
    memcpy(part_message2, message + 512, 4 * 512);
   
    int count = 1;//счётчик для 2ух итераций
   

   
    while (count < 3)
    {
        
        
        uint32_t S1[32]{ 0 };
        uint32_t ch[32]{ 0 };
        uint32_t temp1[32]{ 0 };
        uint32_t S0[32]{ 0 };
        uint32_t maj[32]{ 0 };
        uint32_t temp2[32]{ 0 };

        uint32_t a[32]{ 0 };
        uint32_t b[32]{ 0 };
        uint32_t c[32]{ 0 };
        uint32_t d[32]{ 0 };
        uint32_t e[32]{ 0 };
        uint32_t f[32]{ 0 };
        uint32_t g[32]{ 0 };
        uint32_t h[32]{ 0 };
        uint32_t extend_part_message1[64][32];
        /*for (auto i = 0; i < 64; ++i)
        {
            extend_part_message1[i] = new uint32_t[32]{ 0 };
        }*/

        if (count == 1)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message1 + (i * 32), 4*32);
               // __syncthreads();
            }
        }

        if (count == 2)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message2 + (i * 32), 4*32);
               // __syncthreads();
            }
        }
        
        printf("ext_message_before\n");
        for (auto i = 0; i < 64; ++i)
        {
            for (auto j = 0; j < 32; ++j)
            {
                printf("%d", extend_part_message1[i][j]);
            }
            printf("\n");
        }
        printf("\n");
      
        for (auto i = 16; i < 64; ++i)
        {
            uint32_t s0[32]{ 0 };
            uint32_t s1[32]{ 0 };
            uint32_t sum_ext1_2_s0_s1 = 0;
            //printf("%d", sum_ext1_2_s0_s1);
          
            memcpy(s0, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(extend_part_message1[i - 15], 7)),
                to_binary_32bit(rigth_rotate(extend_part_message1[i - 15], 18)), 32),
                to_binary_32bit(rigth_shift(extend_part_message1[i - 15], 3)), 32), 4 * 32);

            
            
            memcpy(s1, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(extend_part_message1[i - 2], 17)),
                to_binary_32bit(rigth_rotate(extend_part_message1[i - 2], 19)),32),
                to_binary_32bit(rigth_shift(extend_part_message1[i - 2], 10)),32), 4*32);
            
        
            sum_ext1_2_s0_s1 = (sum_strs_32bit(extend_part_message1[i - 16], s0) + sum_strs_32bit(extend_part_message1[i - 7], s1)) % 4294967296;
            
           
            memcpy(extend_part_message1[i], to_binary_32bit(sum_ext1_2_s0_s1), 4*32);
            //__syncthreads();
           // __syncthreads();
        }
        printf("ext_message_after\n");
        for (auto i = 0; i < 64; ++i)
        {
            for (auto j = 0; j < 32; ++j)
            {
                printf("%d", extend_part_message1[i][j]);
            }
            printf("\n");
        }
        printf("\n");
       
        memcpy(a, h0, 4*32);
        memcpy(b, h1, 4*32);
        memcpy(c, h2, 4*32);
        memcpy(d, h3, 4*32);
        memcpy(e, h4, 4*32);
        memcpy(f, h5, 4*32);
        memcpy(g, h6, 4*32);
        memcpy(h, h7, 4*32);
      
        for (auto i = 0; i < 64; ++i)
        {
         
            memcpy(S1, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(e, 6)), to_binary_32bit(rigth_rotate(e, 11)),32),
                to_binary_32bit(rigth_rotate(e, 25)),32), 4*32);
            memcpy(ch, xor_strs(and_strs_32bit(e, f), and_strs_32bit(inverse_str_32bit(e), g),32), 4*32);
           
            memcpy(temp1, to_binary_32bit((sum_strs_32bit(h, S1) + sum_strs_32bit(ch, extend_part_message1[i]) + round_consts[i]) % 4294967296), 4*32);
           
            memcpy(S0, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(a, 2)), to_binary_32bit(rigth_rotate(a, 13)),32),
                to_binary_32bit(rigth_rotate(a, 22)),32), 4*32);
          
            memcpy(maj, xor_strs(xor_strs(and_strs_32bit(a, b), and_strs_32bit(a, c),32),
                and_strs_32bit(b, c),32), 4*32);
          
            memcpy(temp2, to_binary_32bit(sum_strs_32bit(S0, maj)), 4*32);
          
            memcpy(h, g, 4*32);
           
            memcpy(g, f, 4*32);
          
            memcpy(f, e, 4*32);
            memcpy(e, to_binary_32bit(sum_strs_32bit(d, temp1)), 4*32);
            
            memcpy(d, c, 4*32);
            memcpy(c, b, 4*32);
            memcpy(b, a, 4*32);
          
            memcpy(a, to_binary_32bit(sum_strs_32bit(temp1, temp2)), 4*32);
           // __syncthreads();
          //  __syncthreads();
        }
        count++;
       
        memcpy(h0, to_binary_32bit(sum_strs_32bit(h0, a)), 4*32);
        memcpy(h1, to_binary_32bit(sum_strs_32bit(h1, b)), 4*32);
        memcpy(h2, to_binary_32bit(sum_strs_32bit(h2, c)), 4*32);
        memcpy(h3, to_binary_32bit(sum_strs_32bit(h3, d)), 4*32);
        memcpy(h4, to_binary_32bit(sum_strs_32bit(h4, e)), 4*32);
        memcpy(h5, to_binary_32bit(sum_strs_32bit(h5, f)), 4*32);
        memcpy(h6, to_binary_32bit(sum_strs_32bit(h6, g)), 4*32);
        memcpy(h7, to_binary_32bit(sum_strs_32bit(h7, h)), 4*32);
        
        /*delete[] a;
        delete[] b;
        delete[] c;
        delete[] d;
        delete[] e;
        delete[] f;
        delete[] g;
        delete[] h;
        delete[] S0;
        delete[] S1;
        delete[] temp1;
        delete[] temp2;
        delete[] maj;
        delete[] ch;
        delete[] extend_part_message1;*/
     //   __syncthreads();
    }
    
    uint32_t hash[256]{ 0 };
    memcpy(hash, h0, 4*32);
    memcpy(hash + 32, h1, 4*32);
    memcpy(hash + 64, h2, 4*32);
    memcpy(hash + 96, h3, 4*32);
    memcpy(hash + 128, h4, 4*32);
    memcpy(hash + 160, h5, 4*32);
    memcpy(hash + 192, h6, 4*32);
    memcpy(hash + 224, h7, 4*32);
    /*delete[] h0;
    delete[] h1;
    delete[] h2;
    delete[] h3;
    delete[] h4;
    delete[] h5;
    delete[] h6;
    delete[] h7;*/
    memcpy(output_hash, hash, 4 * 256);
  // __syncthreads();
    ///__syncthreads();
    //return output_hash;
    //__syncthreads();
}

__device__ void hmac_sha256( uint32_t* salt, uint32_t* password_xor_with_ipad, uint32_t* password_xor_with_opad)
{
    /*uint32_t *prev_hash = new uint32_t[256]{ 0 };
    uint32_t *hmac_hash = new uint32_t[256]{ 0 };
    memcpy(prev_hash, salt, 4 * 256);
    uint32_t message[1024]{ 0 };
    memcpy(message, preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash), 4 * 1024);
    memcpy(prev_hash, main_loop_sha256(message), 4 * 256);
    memcpy(message, preparation_sha256_with_OPAD(password_xor_with_opad, prev_hash), 4 * 1024);
    memcpy(hmac_hash, main_loop_sha256(message), 4 * 256);*/
   // delete[] prev_hash;
  //  delete[] message;

   // return hmac_hash;
   


}

__device__ void pbkdf2_hmac_sha256(char* password, unsigned int length, unsigned int c)
{
   // 
   // uint32_t *hash = new uint32_t[256]{ 0 };
   // uint32_t *salt = new uint32_t[256]{ 0 };
   // uint32_t *prev_hash = new uint32_t[256]{ 0 };
   // //uint32_t* temp = new uint32_t[256]{ 0 };

   // uint32_t password_xor_with_ipad[512]{ 0 };
   // uint32_t password_xor_with_opad[512]{ 0 };

   // memcpy(password_xor_with_ipad, password_xor_with_IPAD(password, length), 4 * 512);
   // memcpy(password_xor_with_opad, password_xor_with_OPAD(password, length), 4 * 512);
   //// int index = threadIdx.x + blockIdx.x*blockDim.x;
   // uint32_t dklen = 256;
   // uint32_t len = dklen / 256;
   // uint32_t r = dklen - (len - 1) * 256;
   // int index = threadIdx.x;
   //
   // //for (auto index = 0; index < len; ++index)
   // while (index < len)
   // {
   //     salt[255] = index;
   //     memcpy(prev_hash, salt, 4 * 256);
   //     uint32_t temp_hash[256]{ 0 };
   //     for (auto j = 0; j < c; ++j)
   //     {
   //         memcpy(prev_hash, hmac_sha256(prev_hash, password_xor_with_ipad, password_xor_with_opad), 4 * 256);
   //         memcpy(temp_hash, xor_strs(temp_hash, prev_hash,256), 4 * 256);
   //     }
   //    
   //     memcpy(hash + index * 256, temp_hash, 4 * 256);
   // }
   // delete[] salt;
   // delete[] prev_hash;
   // delete[] password_xor_with_ipad;
   // delete[] password_xor_with_opad;
   // return hash;

}

__global__ void preparartion_for_sha256(char* password, unsigned int length,  uint32_t* output_message)
{
    uint32_t* prev_hash = new uint32_t[256]{ 0 };
    //uint32_t* pass_xor_ipad=new uint32_t[512]{ 0 };
    //uint32_t* message = new uint32_t[1024]{ 0 };
    password_xor_with_IPAD(password, length, output_message);
    preparation_sha256_with_IPAD(output_message, prev_hash, output_message);
    
    //__syncthreads();
}

__global__ void addKernel(uint32_t* message, uint32_t* output_hash)
{
    ////int index = threadIdx.x + blockIdx.x;
    //uint32_t* password_xor_with_ipad = new uint32_t[512]{ 0 };
    //uint32_t* password_xor_with_opad = new uint32_t[512]{ 0 };
    //uint32_t* message = new uint32_t[1024]{ 0 };
    //uint32_t* prev_hash = new uint32_t[256]{ 0 };
    //uint32_t* message = new uint32_t[1024]{ 0 };
    ////memcpy(password_xor_with_ipad, password_xor_with_IPAD(password, length), 4 * 512);
    //////memcpy(password_xor_with_opad, password_xor_with_OPAD(password, length), 4 * 512);
    ////memcpy(message, preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash), 4 * 1024);
    ////
    //main_loop_sha256(message, output_hash);
    //__syncthreads();
    main_loop_sha256(message, output_hash);
   // __syncthreads();
    //password_xor_with_IPAD(password, length, output_hash);
    //preparation_sha256_with_IPAD(output_hash, prev_hash, message);
    //main_loop_sha256(message, output_hash);
    //sha256_kernel <<<1, 1>>> (message, output_hash);
}



int main()
{

    std::string password = "1234";
    //char* password = "1234";
    //uint32_t* password_xor_with_ipad = new uint32_t[512]{ 0 };
    //memcpy(password_xor_with_ipad, password_xor_with_IPAD("1234", 4), 4 * 512);
    //uint32_t* prev_hash_u = new uint32_t[256]{ 0 };
    //uint32_t* messsage = new uint32_t[1024]{ 0 };
    //uint32_t* hash = new uint32_t[256]{ 0 };
    //memcpy(hash, main_loop_sha256(preparation_sha256_with_IPAD(password_xor_with_IPAD("1234", 4), prev_hash_u)), 4 * 256);
    //memcpy(messsage, preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash_u), 4 * 1024);
   // memcpy(hash, pbkdf2_hmac_sha256("1234", 4, 4096), 4 * 256);
    //char_to_binary("1234", 4);
   // uint32_t* messsage=preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash_u);
   // std::cout << sizeof(uint32_t) << std::endl;
    uint32_t* hash = new uint32_t[256]{ 0 };
    uint32_t* message_for_sha256=new uint32_t[1024]{ 0 };
    uint32_t* device_hash;
    uint32_t* device_message_1024bits;
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }
        
    

        cudaStatus = hipMalloc((void**)&device_message_1024bits, 1024 * sizeof(uint32_t));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            //goto Error;
        }
        cudaStatus = hipMemcpy(device_message_1024bits, message_for_sha256, 1024 * sizeof(uint32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        

        preparartion_for_sha256 <<<1, 512>>> ("1234", 4, device_message_1024bits);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
            goto Error;
        }

        cudaStatus = hipMemcpy(message_for_sha256, device_message_1024bits, 1024 * sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

       

        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }
//------------------------------------------------------------------------------------------------------------
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&device_hash, 256 * sizeof(uint32_t));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            //goto Error;
        }
        cudaStatus = hipMemcpy(device_hash, hash, 256 * sizeof(uint32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&device_message_1024bits, 1024 * sizeof(uint32_t));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            //goto Error;
        }

        std::cout << "message" << std::endl;
        for (auto i = 0; i < 1024; ++i)
        {
            std::cout << message_for_sha256[i];
        }
        std::cout << std::endl;

        cudaStatus = hipMemcpy(device_message_1024bits, message_for_sha256, 1024 * sizeof(uint32_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        addKernel << <1, 1 >> > (device_message_1024bits, device_hash);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }


        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
            goto Error;
        }

        // Copy output vector from GPU buffer to host memory.
        

        cudaStatus = hipMemcpy(hash, device_hash, 256 * sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
        
        //main_loop_sha256(message_for_sha256, hash);
        for (auto i = 0; i < 256; ++i)
        {
            std::cout << hash[i];
        }
        std::cout << std::endl;

        
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }

        //main_loop_sha256(message_for_sha256, hash);

    //    cudaStatus = hipSetDevice(0);
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    //        goto Error;
    //    }

    //    cudaStatus = hipMalloc((void**)&device_message_1024bits, 1024 * sizeof(uint32_t));
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "hipMalloc failed!");
    //        //goto Error;
    //    }

    //    cudaStatus = hipMemcpy(device_message_1024bits, message_for_sha256, 1024 * sizeof(uint32_t), hipMemcpyHostToDevice);
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "hipMemcpy failed!");
    //        goto Error;
    //    }

    //    cudaStatus = hipMalloc((void**)&device_hash, 256 * sizeof(uint32_t));
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "hipMalloc failed!");
    //        //goto Error;
    //    }
    //    cudaStatus = hipMemcpy(device_hash, hash, 256 * sizeof(uint32_t), hipMemcpyHostToDevice);
    //    if (cudaStatus != hipSuccess) {
    //        fprintf(stderr, "hipMemcpy failed!");
    //        goto Error;
    //    }

    //addKernel <<<1,1>>> (device_message_1024bits,device_hash);
   
    ////Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}
    ////
    ////// hipDeviceSynchronize waits for the kernel to finish, and returns
    ////// any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
    //    goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(hash, device_hash, 256 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //   goto Error;
    //}

    
        
   /* cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/
     //hipDeviceReset must be called before exiting in order for profiling and
     //tracing tools such as Nsight and Visual Profiler to show complete traces.
    


    /*free(device_hash);
    delete[] hash;*/

  
    //// Add vectors in parallel.
     
   

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
   

    return 0;
Error:
    hipFree(device_hash);
    //hipFree(hash);
    hipFree(device_message_1024bits);
    
    //hipFree(dev_b);
}



// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(char* binary_pass, int length_block)
{
    /*int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;*/
    //char* dev_hash_output = "1111";
   // char* dev_binary_pass = "111";
    hipError_t cudaStatus;

//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_binary_pass, length_block *sizeof(char));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//   /* cudaStatus = hipMalloc((void**)&dev_hash_output, length_block * sizeof(char));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }*/
//
//    /*cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }*/
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_binary_pass, binary_pass, length_block * sizeof(char), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//   /* cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }*/
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<16, 16 >>>(dev_binary_pass,length_block);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(binary_pass, dev_binary_pass, length_block * sizeof(char), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_binary_pass);
//   // hipFree(dev_hash_output);
//    //hipFree(dev_b);
    
    return cudaStatus;
}
