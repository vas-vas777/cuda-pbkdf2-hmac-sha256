﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string>
#include <bitset>
#include <iostream>
//#include <openssl/hmac.h>




hipError_t addWithCuda(char* binary_pass, int length_block);

__host__ uint32_t* to_binary_32bit(uint32_t number)
{
    uint32_t *binary_str=new uint32_t [32]{ 0 };


    int count = 31;
    //std::cout << number << std::endl;
    while (number!=0)
    {
       
        binary_str[count] = number % 2;
        
        number /= 2;
        count--;
       
    }
   
    
    return binary_str;
    //delete[] binary_str;
}

__host__ uint32_t str_to_32bitnumber(uint32_t* str)
{
    uint32_t number1 = 0;
    for (auto i = 0; i < 32; ++i)
    {
        number1 += str[i] * pow(2, (31 - i));
    }
    return number1;
}

__host__ uint32_t* and_strs_32bit(uint32_t* str1, uint32_t* str2)
{
    uint32_t *result_str = new uint32_t[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = str1[i] & str2[i];
    }
    return result_str;
    //delete[] result_str;
}

__host__ uint32_t* inverse_str_32bit(uint32_t* str1)
{
    uint32_t *result_str = new uint32_t[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        result_str[i] = !str1[i];
        //std::cout << !str1[i];
    }
   // std::cout << std::endl;
    return result_str;
    //delete[] result_str;
}

__host__ uint32_t* inverse_str_256bit(uint32_t* str1)
{
    uint32_t *result_str = new uint32_t[256]{ 0 };
    for (auto i = 0; i < 256; ++i)
    {
        result_str[i] = !str1[i];
        //std::cout << !str1[i];
    }
    // std::cout << std::endl;
    return result_str;
    //delete[] result_str;
}

__host__ uint32_t sum_strs_32bit(uint32_t* str1, uint32_t* str2)
{
    //uint32_t* result_sum = new uint32_t[32]{ 0 };
    uint32_t number1 = 0;
    uint32_t number2 = 0;
    uint32_t res_number = 0;
    for (auto i = 0; i < 32; ++i)
    {
        number1 += str1[i] * pow(2, (31 - i));
    }
    for (auto i = 0; i < 32; ++i)
    {
        number2 += str2[i] * pow(2, (31 - i));
    }
    //std::cout << number1 << std::endl;
    //std::cout << number2 << std::endl;
    res_number = (number1 + number2) % 4294967296;

    //std::cout << number2 << std::endl;
    return res_number;
   // delete[]result_sum;
    //delete[]result_sum;
}

__host__ uint32_t* xor_strs(uint32_t* str1, uint32_t* str2, unsigned int length)
{
    uint32_t *result_xor=new uint32_t[length];
    for (auto i = 0; i < length; ++i)
    {
        result_xor[i] = str1[i] ^ str2[i];
    }
    /*std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << result_xor[i];
    }
    std::cout << std::endl;*/
    return result_xor;
    //delete[]result_xor;
}

__host__ uint32_t rigth_rotate(uint32_t* str, unsigned int num)
{
    uint32_t number = 0;
    uint32_t *rotated_str = new uint32_t[32]{ 0 };
    memcpy(rotated_str, str, 4 * 32);
    number = str_to_32bitnumber(rotated_str);
    return (number >> num | number << (32 - num));
    
}

__host__ uint32_t rigth_shift(uint32_t* str, unsigned int num)
{
    
    uint32_t shifted_str[32]{ 0 };
    memcpy(shifted_str, str, 4 * 32);
    uint32_t number = str_to_32bitnumber(shifted_str);
    return number >> num;
       
}



__host__ uint32_t* password_xor_with_IPAD(char* password,unsigned int length)
{
    uint32_t *binary_str= new uint32_t[512]{ 0 };
    uint32_t IPAD[] = { 0,0,1,1,0,1,1,0 };
    uint32_t *ipad = new uint32_t[512]{ 0 };
    for (auto i = 0; i < 512; ++i)
    {
        if (i < 8 * length)
        {
           binary_str[i] = (0 != (password[i / 8] & 1 << (~i & 7)));
            ;
        }
        else
        {
            //break;
            binary_str[i] = 0;
        }
    }
    /*std::cout << "password" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;*/
    for (auto i = 0; i < 512; ++i)
    {
        ipad[i] = IPAD[i % 8];
        
    }
    for (auto i = 0; i < 512; ++i)
    {
        binary_str[i] = binary_str[i] ^ ipad[i];
    }
   /* std::cout << "password_xor_with_IPAD 512 bits" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;*/
    //std::cout << sizeof(binary_str) << std::endl;
   // delete[] ipad;
    return binary_str;
  
    
}

__host__ uint32_t* password_xor_with_OPAD(char* password, unsigned int length)
{
    uint32_t *binary_str = new uint32_t[512]{ 0 };
    uint32_t OPAD [] = { 0,1,0,1,1,1,0,0 };
    uint32_t *opad = new uint32_t[512]{ 0 };
    for (auto i = 0; i < 512; ++i)
    {
        if (i < 8 * length)
        {
           // binary_str[i] = (0 != (password[i / 8] & 1 << (~i & 7)));
            ;
        }
        else
        {
            binary_str[i] = 0;
        }
    }
    for (auto i = 0; i < 512; ++i)
    {
        opad[i] = OPAD[i % 8];

    }
   /* for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << opad[i];
    }
    std::cout << std::endl;*/
    for (auto i = 0; i < 512; ++i)
    {
        binary_str[i] = binary_str[i] ^ opad[i];
    }
   /* for (auto i = 0; i < 512; ++i)
    {
        std::cout << binary_str[i];
    }*/

    //delete[] opad;
    return binary_str;
    /*delete[] binary_str;
    delete[] opad;*/
    
}

 __host__ uint32_t* preparation_sha256_with_IPAD(uint32_t* password_xor_with_ipad, uint32_t*prev_hash)
{
     uint32_t *binary_str = new uint32_t[1024]{ 0 };
     memcpy(binary_str, password_xor_with_ipad, 4 * 512);
     memcpy(binary_str + 512, prev_hash, 4 * 256);
    binary_str[768] = 1;
    binary_str[1014] = 1;
    binary_str[1015] = 1;
    return binary_str;
}

__host__ uint32_t* preparation_sha256_with_OPAD(uint32_t* password_xor_with_opad, uint32_t* prev_hash)
{
    uint32_t binary_str[1024]{ 0 };
  

    memcpy(binary_str, password_xor_with_opad, 4 * 512);
    memcpy(binary_str + 512, prev_hash, 4 * 256);
    binary_str[768] = 1;
    binary_str[1014] = 1;
    binary_str[1015] = 1;
   
    return binary_str;
   
}

__host__ uint32_t* main_loop_sha256(uint32_t* message)
{
    const uint32_t round_consts[64] = { 0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

    uint32_t* h0 = new uint32_t[] { 0, 1, 1, 0,  1, 0, 1, 0,  0, 0, 0, 0,  1, 0, 0, 1,  1, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 0,  0, 1, 1, 1 };
    uint32_t* h1 = new uint32_t[] { 1, 0, 1, 1,  1, 0, 1, 1,  0, 1, 1, 0,  0, 1, 1, 1,  1, 0, 1, 0,  1, 1, 1, 0,  1, 0, 0, 0,  0, 1, 0, 1 };
    uint32_t* h2 = new uint32_t[] { 0, 0, 1, 1,  1, 1, 0, 0,  0, 1, 1, 0,  1, 1, 1, 0,  1, 1, 1, 1,  0, 0, 1, 1,  0, 1, 1, 1,  0, 0, 1, 0 };
    uint32_t* h3 = new uint32_t[] { 1, 0, 1, 0,  0, 1, 0, 1,  0, 1, 0, 0,  1, 1, 1, 1,  1, 1, 1, 1,  0, 1, 0, 1,  0, 0, 1, 1,  1, 0, 1, 0 };
    uint32_t* h4 = new uint32_t[] { 0, 1, 0, 1,  0, 0, 0, 1,  0, 0, 0, 0,  1, 1, 1, 0,  0, 1, 0, 1,  0, 0, 1, 0,  0, 1, 1, 1,  1, 1, 1, 1 };
    uint32_t* h5 = new uint32_t[] { 1, 0, 0, 1,  1, 0, 1, 1,  0, 0, 0, 0,  0, 1, 0, 1,  0, 1, 1, 0,  1, 0, 0, 0,  1, 0, 0, 0,  1, 1, 0, 0 };
    uint32_t* h6 = new uint32_t[] { 0, 0, 0, 1,  1, 1, 1, 1,  1, 0, 0, 0,  0, 0, 1, 1,  1, 1, 0, 1,  1, 0, 0, 1,  1, 0, 1, 0,  1, 0, 1, 1 };
    uint32_t* h7 = new uint32_t[] { 0, 1, 0, 1,  1, 0, 1, 1,  1, 1, 1, 0,  0, 0, 0, 0,  1, 1, 0, 0,  1, 1, 0, 1,  0, 0, 0, 1,  1, 0, 0, 1 };
    uint32_t part_message1[512]{ 0 };/*{ 0,1,1,0,1,0,0,0, 0,1,1,0,0,1,0,1, 0,1,1,0,1,1,0,0, 0,1,1,0,1,1,0,0, 0,1,1,0,1,1,1,1, 0,0,1,0,0,0,0,0, 0,1,1,1,0,1,1,1, 0,1,1,0,1,1,1,1,
0,1,1,1,0,0,1,0, 0,1,1,0,1,1,0,0, 0,1,1,0,0,1,0,0, 1,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,0,0,0,0,0,0,0, 0,1,0,1,1,0,0,0 };*/
    uint32_t part_message2[512]{ 0 };
    memcpy(part_message1, message, 4 * 512);
    memcpy(part_message2, message + 512, 4 * 512);
   /* std::cout << "message" << std::endl;
    for (auto i = 0; i < 1024; ++i)
    {
        std::cout << message[i];
    }
    std::cout << std::endl;*/
   /* std::cout << "part_message1" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << part_message1[i];
    }
    std::cout << std::endl;
    std::cout << std::endl;
    std::cout << "part_message2" << std::endl;
    for (auto i = 0; i < 512; ++i)
    {
        std::cout << part_message2[i];
    }
    std::cout << std::endl;*/
    int count = 1;//счётчик для 2ух итераций
   

   
    while (count < 3)
    {
        
        
        uint32_t *S1 = new uint32_t[32]{ 0 };
        uint32_t *ch = new uint32_t[32]{ 0 };
        uint32_t *temp1 = new uint32_t[32]{ 0 };
        uint32_t *S0 = new uint32_t[32]{ 0 };
        uint32_t *maj = new uint32_t[32]{ 0 };
        uint32_t *temp2 = new uint32_t[32]{ 0 };

        
        //uint32_t sum_ext2_s1 = 0;

        uint32_t *a = new uint32_t[32]{ 0 };
        uint32_t *b = new uint32_t[32]{ 0 };
        uint32_t *c = new uint32_t[32]{ 0 };
        uint32_t *d = new uint32_t[32]{ 0 };
        uint32_t *e = new uint32_t[32]{ 0 };
        uint32_t *f = new uint32_t[32]{ 0 };
        uint32_t *g = new uint32_t[32]{ 0 };
        uint32_t *h = new uint32_t[32]{ 0 };
        uint32_t extend_part_message1[64][32];// = new uint32_t * [64];
        /*for (auto i = 0; i < 64; ++i)
        {
            extend_part_message1[i] = new uint32_t[32]{ 0 };
        }*/

        if (count == 1)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message1 + (i * 32), 4*32);
            }
        }

        if (count == 2)
        {
            for (auto i = 0; i < 16; ++i)
            {
                memcpy(extend_part_message1[i], part_message2 + (i * 32), 4*32);
            }
        }
       /* std::cout << "extend_part_message" << std::endl;
        for (auto i = 0; i < 64; ++i)
        {
            std::cout << i << "-";
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;*/

        for (auto i = 16; i < 64; ++i)
        {
            uint32_t *s0 = new uint32_t[32]{ 0 };
            uint32_t *s1 = new uint32_t[32]{ 0 };
            uint32_t sum_ext1_2_s0_s1 = 0;
            //std::cout << "start-i=" <<i<<"i-15="<<i-15<< std::endl;
            memcpy(s0, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(extend_part_message1[i - 15], 7)),
                to_binary_32bit(rigth_rotate(extend_part_message1[i - 15], 18)), 32),
                to_binary_32bit(rigth_shift(extend_part_message1[i - 15], 3)), 32), 4 * 32);
            /*std::cout << "s0 - " << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << s0[j];
            }
            std::cout << std::endl;
            std::cout << "start2-i=" << i << "i-2=" << i - 2 << std::endl;*/
            memcpy(s1, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(extend_part_message1[i - 2], 17)),
                to_binary_32bit(rigth_rotate(extend_part_message1[i - 2], 19)),32),
                to_binary_32bit(rigth_shift(extend_part_message1[i - 2], 10)),32), 4*32);
            /*std::cout << "s1 - " << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << s1[j];
            }
            std::cout << std::endl;
            std::cout << "exte[i-16] - i-16="<<i-16 << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i-16][j];
            }
            std::cout << std::endl;
            std::cout << "exte[i-7] - i-7=" << i - 7 << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i - 7][j];
            }
            std::cout << std::endl;*/
        
            sum_ext1_2_s0_s1 = (sum_strs_32bit(extend_part_message1[i - 16], s0) + sum_strs_32bit(extend_part_message1[i - 7], s1)) % 4294967296;
            //std::cout << "i-" << i << " " << sum_ext1_2_s0_s1 << std::endl;
            //std::cout << std::endl;
           
            memcpy(extend_part_message1[i], to_binary_32bit(sum_ext1_2_s0_s1), 4*32);
           /* std::cout << "exte[" << i << "]" << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << j;
            }
            std::cout << std::endl;
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;*/
            //delete[] s0;
            //delete[] s1;

        }
        /*std::cout << "extend_part_message_after" << std::endl;
        for (auto i = 0; i < 64; ++i)
        {
            std::cout << i << "- ";
            for (auto j = 0; j < 32; ++j)
            {
                std::cout << extend_part_message1[i][j];
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;*/
       
        memcpy(a, h0, 4*32);
        memcpy(b, h1, 4*32);
        memcpy(c, h2, 4*32);
        memcpy(d, h3, 4*32);
        memcpy(e, h4, 4*32);
        memcpy(f, h5, 4*32);
        memcpy(g, h6, 4*32);
        memcpy(h, h7, 4*32);
      
        for (auto i = 0; i < 64; ++i)
        {
         
            memcpy(S1, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(e, 6)), to_binary_32bit(rigth_rotate(e, 11)),32),
                to_binary_32bit(rigth_rotate(e, 25)),32), 4*32);
            memcpy(ch, xor_strs(and_strs_32bit(e, f), and_strs_32bit(inverse_str_32bit(e), g),32), 4*32);
           
            memcpy(temp1, to_binary_32bit((sum_strs_32bit(h, S1) + sum_strs_32bit(ch, extend_part_message1[i]) + round_consts[i]) % 4294967296), 4*32);
           
            memcpy(S0, xor_strs(xor_strs(to_binary_32bit(rigth_rotate(a, 2)), to_binary_32bit(rigth_rotate(a, 13)),32),
                to_binary_32bit(rigth_rotate(a, 22)),32), 4*32);
          
            memcpy(maj, xor_strs(xor_strs(and_strs_32bit(a, b), and_strs_32bit(a, c),32),
                and_strs_32bit(b, c),32), 4*32);
          
            memcpy(temp2, to_binary_32bit(sum_strs_32bit(S0, maj)), 4*32);
          
            memcpy(h, g, 4*32);
           
            memcpy(g, f, 4*32);
          
            memcpy(f, e, 4*32);
            memcpy(e, to_binary_32bit(sum_strs_32bit(d, temp1)), 4*32);
            
            memcpy(d, c, 4*32);
            memcpy(c, b, 4*32);
            memcpy(b, a, 4*32);
          
            memcpy(a, to_binary_32bit(sum_strs_32bit(temp1, temp2)), 4*32);
           
        }
        count++;
       
        memcpy(h0, to_binary_32bit(sum_strs_32bit(h0, a)), 4*32);
        memcpy(h1, to_binary_32bit(sum_strs_32bit(h1, b)), 4*32);
        memcpy(h2, to_binary_32bit(sum_strs_32bit(h2, c)), 4*32);
        memcpy(h3, to_binary_32bit(sum_strs_32bit(h3, d)), 4*32);
        memcpy(h4, to_binary_32bit(sum_strs_32bit(h4, e)), 4*32);
        memcpy(h5, to_binary_32bit(sum_strs_32bit(h5, f)), 4*32);
        memcpy(h6, to_binary_32bit(sum_strs_32bit(h6, g)), 4*32);
        memcpy(h7, to_binary_32bit(sum_strs_32bit(h7, h)), 4*32);
        /*delete[] a;
        delete[] b;
        delete[] c;
        delete[] d;
        delete[] e;
        delete[] f;
        delete[] g;
        delete[] h;
        delete[] S0;
        delete[] S1;
        delete[] temp1;
        delete[] temp2;
        delete[] maj;
        delete[] ch;*/


               
    }
    /*std::cout << "h0" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    std::cout << "h1" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h1[i];
    }
    std::cout << std::endl;
    std::cout << "h2" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h2[i];
    }
    std::cout << std::endl;
    std::cout << "h3" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h3[i];
    }
    std::cout << std::endl;
    std::cout << "h4" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h4[i];
    }
    std::cout << std::endl;
    std::cout << "h5" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h5[i];
    }
    std::cout << std::endl;
    std::cout << "h6" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h6[i];
    }
    std::cout << std::endl;
    std::cout << "h7" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h7[i];
    }
    std::cout << std::endl;*/
    uint32_t *hash = new uint32_t[256]{ 0 };
    memcpy(hash, h0, 4*32);
    memcpy(hash + 32, h1, 4*32);
    memcpy(hash + 64, h2, 4*32);
    memcpy(hash + 96, h3, 4*32);
    memcpy(hash + 128, h4, 4*32);
    memcpy(hash + 160, h5, 4*32);
    memcpy(hash + 192, h6, 4*32);
    memcpy(hash + 224, h7, 4*32);
    /*delete[] h0;
    delete[] h1;
    delete[] h2;
    delete[] h3;
    delete[] h4;
    delete[] h5;
    delete[] h6;
    delete[] h7;*/

    return hash;
}

__host__ uint32_t* hmac_sha256( uint32_t* salt, uint32_t* password_xor_with_ipad, uint32_t* password_xor_with_opad)
{
    uint32_t *prev_hash = new uint32_t[256]{ 0 };
    uint32_t *hmac_hash = new uint32_t[256]{ 0 };
    memcpy(prev_hash, salt, 4 * 256);
    uint32_t message[1024]{ 0 };
    memcpy(message, preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash), 4 * 1024);
    memcpy(prev_hash, main_loop_sha256(message), 4 * 256);
    memcpy(message, preparation_sha256_with_OPAD(password_xor_with_opad, prev_hash), 4 * 1024);
    memcpy(hmac_hash, main_loop_sha256(message), 4 * 256);
   // delete[] prev_hash;
  //  delete[] message;

    return hmac_hash;
   


}

__host__ uint32_t* pbkdf2_hmac_sha256(char* password, unsigned int length, unsigned int c)
{
    
    uint32_t *hash = new uint32_t[256]{ 0 };
    uint32_t *salt = new uint32_t[256]{ 0 };
    uint32_t *prev_hash = new uint32_t[256]{ 0 };
    //uint32_t* temp = new uint32_t[256]{ 0 };

    uint32_t password_xor_with_ipad[512]{ 0 };
    uint32_t password_xor_with_opad[512]{ 0 };

    memcpy(password_xor_with_ipad, password_xor_with_IPAD(password, length), 4 * 512);
    memcpy(password_xor_with_opad, password_xor_with_OPAD(password, length), 4 * 512);
   // int index = threadIdx.x + blockIdx.x*blockDim.x;
    uint32_t dklen = 256;
    uint32_t len = dklen / 256;
    uint32_t r = dklen - (len - 1) * 256;
   
    for (auto index = 0; index < len; ++index)
    {
        salt[255] = index;
        memcpy(prev_hash, salt, 4 * 256);
        uint32_t temp_hash[256]{ 0 };
        for (auto j = 0; j < c; ++j)
        {
            memcpy(prev_hash, hmac_sha256(prev_hash, password_xor_with_ipad, password_xor_with_opad), 4 * 256);
            memcpy(temp_hash, xor_strs(temp_hash, prev_hash,256), 4 * 256);
        }
       
        memcpy(hash + index * 256, temp_hash, 4 * 256);
    }
    /*delete[] salt;
    delete[] prev_hash;
    delete[] password_xor_with_ipad;
    delete[] password_xor_with_opad;*/
    return hash;

}

__global__ void addKernel(char* password, unsigned int length, unsigned int c, uint32_t* output_hash)
{
    ////int index = threadIdx.x + blockIdx.x;
    //uint32_t* password_xor_with_ipad = new uint32_t[512]{ 0 };
    //uint32_t* password_xor_with_opad = new uint32_t[512]{ 0 };
    //uint32_t* message = new uint32_t[1024]{ 0 };
    //uint32_t* prev_hash = new uint32_t[256]{ 0 };
    //memcpy(password_xor_with_ipad, password_xor_with_IPAD(password, length), 4 * 512);
    ////memcpy(password_xor_with_opad, password_xor_with_OPAD(password, length), 4 * 512);
    //memcpy(message, preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash), 4 * 1024);
    //


    //memcpy(output_hash, main_loop_sha256(message), 4 * 256);
   /* delete[] message;
    delete[] prev_hash;
    delete[] password_xor_with_ipad;
    delete[] password_xor_with_opad;*/
}
int main()
{
   
    std::string password = "1234";
    //char* password = "1234";
    //uint32_t* password_xor_with_ipad = new uint32_t[512]{ 0 };
    //memcpy(password_xor_with_ipad, password_xor_with_IPAD("1234", 4), 4 * 512);
    //uint32_t* prev_hash_u = new uint32_t[256]{ 0 };
    //uint32_t* messsage = new uint32_t[1024]{ 0 };
    uint32_t* hash = new uint32_t[256]{ 0 };

    //memcpy(messsage, preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash_u), 4 * 1024);
    memcpy(hash, pbkdf2_hmac_sha256("1234", 4, 4096), 4 * 256);
    std::cout << "hash" << std::endl;
    for (auto i = 0; i < 256; ++i)
    {
        std::cout << hash[i];
    }
   // uint32_t* messsage=preparation_sha256_with_IPAD(password_xor_with_ipad, prev_hash_u);
    //std::cout << sizeof(uint32_t) << std::endl;
    //uint32_t* hash = new uint32_t[256]{ 0 };
    //uint32_t* device_hash;
    //cudaError_t cudaStatus;
    //cudaStatus = cudaSetDevice(0);
    //    if (cudaStatus != cudaSuccess) {
    //        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    //        //goto Error;
    //    }
    //    //cudaStatus = cudaMalloc((void**)&hash, 256 * sizeof(uint32_t));
    //    //if (cudaStatus != cudaSuccess) {
    //    //    fprintf(stderr, "cudaMalloc failed!");
    //    //    //goto Error;
    //    //}
    //cudaStatus=cudaMalloc((void**)&device_hash, 256* sizeof(uint32_t));
    //    if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMalloc failed!");
    //    //goto Error;
    //}
    //    cudaStatus = cudaMemcpy(device_hash, hash, 256*sizeof(uint32_t),cudaMemcpyHostToDevice);
    //    if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMemcpy failed!");
    //    //goto Error;
    //}
    //addKernel <<<4,512>>> ("1234", 4, 10000, device_hash);
   
    ////Check for any errors launching the kernel
    //cudaStatus = cudaGetLastError();
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
    // //   goto Error;
    //}
    //
    //// cudaDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = cudaDeviceSynchronize();
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel %s!\n", cudaStatus, cudaGetErrorString(cudaStatus));
    //   // goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = cudaMemcpy(hash, device_hash, 256 * sizeof(uint32_t), cudaMemcpyDeviceToHost);
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMemcpy failed!");
    //   // goto Error;
    //}
    //cudaDeviceSynchronize();
    //uint32_t* fin_hash = new uint32_t[4*256]{ 0 };
    //memcpy(fin_hash, hash, 256 * 4);
    //cudaDeviceReset();
    //memcpy(hash, pbkdf2_hmac_sha256("1234", 4, 400),4 * 256);
    //uint32_t* hash=main_loop_sha256_with_ipad(messsage);
    
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
   /* cudaStatus = cudaDeviceReset();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceReset failed!");
        return 1;
    }*/
    //free(device_hash);
   // delete[] hash;

  /* uint32_t* h0 = new uint32_t[] { 1, 0, 0, 0, 0, 1, 1, 0, 1, 1, 0, 1, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 1 };
   uint32_t* res = new uint32_t[32]{ 0 };
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    std::cout << sizeof(uint32_t) << std::endl;
    memcpy(res, to_binary_32bit(rigth_rotate(h0, 10)), 4 * 32);
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << res[i];
    }
    std::cout << std::endl;
    std::cout << rigth_rotate(h0, 10) << std::endl;*/
   /* uint32_t* h0 = new uint32_t[] { 0, 1, 1, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1 };
    uint32_t* h1 = new uint32_t[] { 1, 0, 1, 1, 1, 0, 1, 1, 0, 1, 1, 0, 0, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 1, 0, 1 };
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h1[i];
    }
    std::cout << std::endl;
    uint32_t* prev_hash_u;
    uint32_t number = 0;
    prev_hash_u = rigth_rotate(h0, 32);
    std::cout << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << prev_hash_u[i];
    }*/
   // std::wcout << number << std::endl;
   
    /*
    uint32_t* h0 = new uint32_t[] { 0, 1, 1, 0, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1 };
    uint32_t* h1 = new uint32_t[] { 1, 0, 1, 1, 1, 0, 1, 1, 0, 1, 1, 0, 0, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 0, 0, 1, 0, 1 };
    uint32_t* res = new uint32_t[32];
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << h0[i];
    }
    std::cout << std::endl;
    res = rigth_shift(h0, 5);
    
    std::cout << "res" << std::endl;
    for (auto i = 0; i < 32; ++i)
    {
        std::cout << res[i];
    }*/
    //password_xor_with_IPAD("1234", 4);
    // Add vectors in parallel.
    /*cudaError_t cudaStatus = addWithCuda(bin_str_512, 512);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }*/

   /* printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);*/
   

    //// cudaDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = cudaDeviceReset();
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(char* binary_pass, int length_block)
{
    /*int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;*/
    //char* dev_hash_output = "1111";
   // char* dev_binary_pass = "111";
    hipError_t cudaStatus;

//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = cudaSetDevice(0);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = cudaMalloc((void**)&dev_binary_pass, length_block *sizeof(char));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//   /* cudaStatus = cudaMalloc((void**)&dev_hash_output, length_block * sizeof(char));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }*/
//
//    /*cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }*/
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = cudaMemcpy(dev_binary_pass, binary_pass, length_block * sizeof(char), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//   /* cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }*/
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<16, 16 >>>(dev_binary_pass,length_block);
//
//    // Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(binary_pass, dev_binary_pass, length_block * sizeof(char), cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    cudaFree(dev_binary_pass);
//   // cudaFree(dev_hash_output);
//    //cudaFree(dev_b);
    
    return cudaStatus;
}
